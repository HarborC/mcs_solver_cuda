#include "hip/hip_runtime.h"
#include "common_cuda.h"
#include <stdio.h> 
#include "qr_cuda.h"

__device__ int find_first_nonzero(Eigen::Matrix<double,1,165>& a)
{
    int idx = -1;
    for (int i = 0; i < 165; i++)
    {
        if (abs(a(i))>NEAR_ZERO_THRESHOLD)
        {
            idx = i;
            break;
        }
    }
    return idx;
}

__device__ int find_column(Eigen::Matrix<int,3,165>& a, Eigen::Matrix<int,3,1>& b)
{
    int idx = -1;
    for (int i = 0; i < 165; i++)
    {
        if (a(0,i)==b(0) && a(1,i)==b(1) && a(2,i)==b(2))
        {
            idx = i;
            break;
        }
    }
    return idx;
}

__device__ int find_column(Eigen::Matrix<int,3,84>& a, Eigen::Matrix<int,3,1>& b)
{
    int idx = -1;
    for (int i = 0; i < 84; i++)
    {
        if (a(0,i)==b(0) && a(1,i)==b(1) && a(2,i)==b(2))
        {
            idx = i;
            break;
        }
    }
    return idx;
}

__device__ void quot_var3_order8_by_x2y2z2_1(Eigen::Matrix<double,1,165>& c, Eigen::Matrix<double,1,84>& c_quot)
{
    Eigen::Matrix<double,1,165> c1 = c;
    c_quot.setZero();
    Eigen::Matrix<int,3,165> M1;
    Eigen::Matrix<int,3,4> M2;
    Eigen::Matrix<int,3,84> M_quot;
    M1 << 
        8, 7, 7, 7, 6, 6, 6, 6, 6, 6, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
        0, 1, 0, 0, 2, 1, 1, 0, 0, 0, 3, 2, 2, 1, 1, 1, 0, 0, 0, 0, 4, 3, 3, 2, 2, 2, 1, 1, 1, 1, 0, 0, 0, 0, 0, 5, 4, 4, 3, 3, 3, 2, 2, 2, 2, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 6, 5, 5, 4, 4, 4, 3, 3, 3, 3, 2, 2, 2, 2, 2, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 7, 6, 6, 5, 5, 5, 4, 4, 4, 4, 3, 3, 3, 3, 3, 2, 2, 2, 2, 2, 2, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 8, 7, 7, 6, 6, 6, 5, 5, 5, 5, 4, 4, 4, 4, 4, 3, 3, 3, 3, 3, 3, 2, 2, 2, 2, 2, 2, 2, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
        0, 0, 1, 0, 0, 1, 0, 2, 1, 0, 0, 1, 0, 2, 1, 0, 3, 2, 1, 0, 0, 1, 0, 2, 1, 0, 3, 2, 1, 0, 4, 3, 2, 1, 0, 0, 1, 0, 2, 1, 0, 3, 2, 1, 0, 4, 3, 2, 1, 0, 5, 4, 3, 2, 1, 0, 0, 1, 0, 2, 1, 0, 3, 2, 1, 0, 4, 3, 2, 1, 0, 5, 4, 3, 2, 1, 0, 6, 5, 4, 3, 2, 1, 0, 0, 1, 0, 2, 1, 0, 3, 2, 1, 0, 4, 3, 2, 1, 0, 5, 4, 3, 2, 1, 0, 6, 5, 4, 3, 2, 1, 0, 7, 6, 5, 4, 3, 2, 1, 0, 0, 1, 0, 2, 1, 0, 3, 2, 1, 0, 4, 3, 2, 1, 0, 5, 4, 3, 2, 1, 0, 6, 5, 4, 3, 2, 1, 0, 7, 6, 5, 4, 3, 2, 1, 0, 8, 7, 6, 5, 4, 3, 2, 1, 0;
    M2 << 
        2, 0, 0, 0,
        0, 2, 0, 0,
        0, 0, 2, 0;
    M_quot <<
        6, 5, 4, 3, 2, 1, 0, 5, 4, 3, 2, 1, 0, 4, 3, 2, 1, 0, 3, 2, 1, 0, 2, 1, 0, 1, 0, 0, 5, 4, 3, 2, 1, 0, 4, 3, 2, 1, 0, 3, 2, 1, 0, 2, 1, 0, 1, 0, 0, 4, 3, 2, 1, 0, 3, 2, 1, 0, 2, 1, 0, 1, 0, 0, 3, 2, 1, 0, 2, 1, 0, 1, 0, 0, 2, 1, 0, 1, 0, 0, 1, 0, 0, 0, 
        0, 1, 2, 3, 4, 5, 6, 0, 1, 2, 3, 4, 5, 0, 1, 2, 3, 4, 0, 1, 2, 3, 0, 1, 2, 0, 1, 0, 0, 1, 2, 3, 4, 5, 0, 1, 2, 3, 4, 0, 1, 2, 3, 0, 1, 2, 0, 1, 0, 0, 1, 2, 3, 4, 0, 1, 2, 3, 0, 1, 2, 0, 1, 0, 0, 1, 2, 3, 0, 1, 2, 0, 1, 0, 0, 1, 2, 0, 1, 0, 0, 1, 0, 0, 
        0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 2, 2, 2, 2, 2, 3, 3, 3, 3, 4, 4, 4, 5, 5, 6, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 2, 2, 2, 2, 3, 3, 3, 4, 4, 5, 0, 0, 0, 0, 0, 1, 1, 1, 1, 2, 2, 2, 3, 3, 4, 0, 0, 0, 0, 1, 1, 1, 2, 2, 3, 0, 0, 0, 1, 1, 2, 0, 0, 1, 0;
    Eigen::Matrix<double,1,4> c2;
    c2 << 1, 1, 1, 1;
    
    Eigen::Matrix<int,3,1> LMid, m;
    Eigen::Matrix<int,3,1> LMd = M2.col(0);
    while(true)
    {
        int idx = find_first_nonzero(c1);
        if (idx==-1)
            break;

        LMid = M1.col(idx);
        bool divisible = false;

        m = LMid - LMd;
        if (m(0)>=0 && m(1)>=0 && m(2)>=0)
        {
            double c = c1(idx)/c2(0);
            int idx2 = find_column(M_quot, m);
            if (idx2==-1)
            {
                continue;
            }
            c_quot(idx2) = c_quot(idx2) + c;
            for (int k = 0; k < 4; k++)
            {
                Eigen::Matrix<int,3,1> m_tmp = m + M2.col(k);
                double c_tmp = c*c2(k);
                int idx3 = find_column(M1, m_tmp);
                if (idx3==-1)
                {
                    continue;
                }
                c1(idx3) =  c1(idx3) - c_tmp;
            }
            divisible = true;
        }
        if (!divisible)
        {
            int idx = find_first_nonzero(c1);
            if (idx==-1)
                break;
            c1(idx) = 0;
        }
    }
    return;
}

__device__ void construct_order6_poly(Eigen::Matrix<double,1,10>** M, 
    Eigen::Matrix<double,15,84>& C, Eigen::Matrix<double,15,165>& C8, Eigen::MatrixXi& Sub_index) {
    C.setZero();
    C8.setZero();

    Eigen::Matrix<int,15,4> Idx_all;
    Idx_all << 
     0, 1, 2, 3, 
     0, 1, 2, 4, 
     0, 1, 2, 5,
     0, 1, 3, 4,
     0, 1, 3, 5,
     0, 1, 4, 5,
     0, 2, 3, 4,
     0, 2, 3, 5,
     0, 2, 4, 5,
     0, 3, 4, 5,
     1, 2, 3, 4,
     1, 2, 3, 5,
     1, 2, 4, 5,
     1, 3, 4, 5,
     2, 3, 4, 5;

    int N = Sub_index.cols()*Sub_index.rows();
    
    for (int k = 0; k < N; k++)
    {
        int i = Sub_index(k);
        Eigen::Matrix<int,1,4> idx = Idx_all.row(i);

        Eigen::Matrix<double,1,10> 
            m11, m12, m13, m14, 
            m21, m22, m23, m24,
            m31, m32, m33, m34,
            m41, m42, m43, m44;

        m11 = M[idx(0)][0]; m12 = M[idx(0)][1]; m13 = M[idx(0)][2], m14 = M[idx(0)][3];
        m21 = M[idx(1)][0]; m22 = M[idx(1)][1]; m23 = M[idx(1)][2], m24 = M[idx(1)][3];
        m31 = M[idx(2)][0]; m32 = M[idx(2)][1]; m33 = M[idx(2)][2], m34 = M[idx(2)][3];
        m41 = M[idx(3)][0]; m42 = M[idx(3)][1]; m43 = M[idx(3)][2], m44 = M[idx(3)][3];

        Eigen::Matrix<double,1,165> d, d0;
        d.setZero();
        d0.setZero();
        var3_order2_four_multiplication(m11, m22, m33, m44, d0); d = d + d0;
        var3_order2_four_multiplication(m11, m22, m34, m43, d0); d = d - d0;
        var3_order2_four_multiplication(m11, m23, m32, m44, d0); d = d - d0;
        var3_order2_four_multiplication(m11, m23, m34, m42, d0); d = d + d0;
        var3_order2_four_multiplication(m11, m24, m32, m43, d0); d = d + d0;
        var3_order2_four_multiplication(m11, m24, m33, m42, d0); d = d - d0;
        var3_order2_four_multiplication(m12, m21, m33, m44, d0); d = d - d0;
        var3_order2_four_multiplication(m12, m21, m34, m43, d0); d = d + d0;
        var3_order2_four_multiplication(m12, m23, m31, m44, d0); d = d + d0;
        var3_order2_four_multiplication(m12, m23, m34, m41, d0); d = d - d0;
        var3_order2_four_multiplication(m12, m24, m31, m43, d0); d = d - d0;
        var3_order2_four_multiplication(m12, m24, m33, m41, d0); d = d + d0;
        var3_order2_four_multiplication(m13, m21, m32, m44, d0); d = d + d0;
        var3_order2_four_multiplication(m13, m21, m34, m42, d0); d = d - d0;
        var3_order2_four_multiplication(m13, m22, m31, m44, d0); d = d - d0;
        var3_order2_four_multiplication(m13, m22, m34, m41, d0); d = d + d0;
        var3_order2_four_multiplication(m13, m24, m31, m42, d0); d = d + d0;
        var3_order2_four_multiplication(m13, m24, m32, m41, d0); d = d - d0;
        var3_order2_four_multiplication(m14, m21, m32, m43, d0); d = d - d0;
        var3_order2_four_multiplication(m14, m21, m33, m42, d0); d = d + d0;
        var3_order2_four_multiplication(m14, m22, m31, m43, d0); d = d + d0;
        var3_order2_four_multiplication(m14, m22, m33, m41, d0); d = d - d0;
        var3_order2_four_multiplication(m14, m23, m31, m42, d0); d = d - d0;
        var3_order2_four_multiplication(m14, m23, m32, m41, d0); d = d + d0;

        Eigen::Matrix<double,1,84> quot;
        quot_var3_order8_by_x2y2z2_1(d, quot);
        C.block(i, 0, 1, 84) = quot;
        C8.block(i, 0, 1, 165) = d;
    }
    return;
}

__device__ void construct_order6_extra_poly(Eigen::Matrix<double,1,10>** M, 
    Eigen::Matrix<double,2,84>& C, Eigen::Matrix<double,2,165>& C8, Eigen::MatrixXi Sub_index_extra) {
    C.setZero();
    C8.setZero();

    Eigen::Matrix<int,2,3> Idx_all;
    Idx_all << 
         0, 1, 2,
         3, 4, 5;

    Eigen::Matrix<double,1,10> m_one;
    m_one.setZero(); m_one(9) = 1;

    int N = Sub_index_extra.cols()*Sub_index_extra.rows();
    for (int k = 0; k < N; k++)
    {
        int i = Sub_index_extra(k);
        Eigen::Matrix<int,1,3> idx = Idx_all.row(i);

        Eigen::Matrix<double,1,10> m11, m12, m13, 
                                   m21, m22, m23,
                                   m31, m32, m33,
                                   m44;

        m11 = M[idx(0)][0]; m12 = M[idx(0)][1]; m13 = M[idx(0)][2];
        m21 = M[idx(1)][0]; m22 = M[idx(1)][1]; m23 = M[idx(1)][2];
        m31 = M[idx(2)][0]; m32 = M[idx(2)][1]; m33 = M[idx(2)][2];
        m44 = m_one;

        Eigen::Matrix<double,1,165> d, d0;
        d.setZero();
        d0.setZero();
        var3_order2_four_multiplication(m11, m22, m33, m44, d0); d = d + d0;
        var3_order2_four_multiplication(m11, m23, m32, m44, d0); d = d - d0;
        var3_order2_four_multiplication(m12, m21, m33, m44, d0); d = d - d0;
        var3_order2_four_multiplication(m12, m23, m31, m44, d0); d = d + d0;
        var3_order2_four_multiplication(m13, m21, m32, m44, d0); d = d + d0;
        var3_order2_four_multiplication(m13, m22, m31, m44, d0); d = d - d0;

        Eigen::Matrix<double,1,84> quot;
        quot_var3_order8_by_x2y2z2_1(d, quot);
        C.block(i, 0, 1, 84) = quot;
        C8.block(i, 0, 1, 165) = d;
    }
    return;
}

__device__ void construct_M(
    Eigen::Matrix<double,1,10>** M, 
    Eigen::Matrix3d* R_camera, Eigen::Vector3d* T_camera, 
    Eigen::Matrix3d* Ac, Eigen::Vector3d* Image1, Eigen::Vector3d* Image2,
    AC_TYPE actype, bool is_known_angle) {

    int point_num = 2;
    Eigen::Matrix<double,2,10> 
        f1_New_C11, f1_New_C12, f1_New_C13, f1_New_C14,
        f2_New_C21, f2_New_C22, f2_New_C23, f2_New_C24,
        f3_New_C31, f3_New_C32, f3_New_C33, f3_New_C34;
    for (int i = 0; i < point_num; i++)
    {
        Eigen::Vector3d P1 = Image1[i];
        Eigen::Vector3d P2 = Image2[i];
        Eigen::Vector3d U1 = P1;
        U1.normalize();
        Eigen::Vector3d U2 = P2;
        U2.normalize();

        int idx1 = 0;
        int idx2 = 0;
		if(actype == GENERIC_CONSTRAINT_FULL || actype == GENERIC_CONSTRAINT_PARTIAL)
        {
            if (i==0)
            {
                idx1 = 0;
                idx2 = 1;
            }
            else
            {
                idx1 = 2;
                idx2 = 3;
            }
        }
        else if (actype == CASE5_CONSTRAINT_FULL || actype == CASE5_CONSTRAINT_PARTIAL)
        {
            if (i==0)
            {
                idx1 = 0;
                idx2 = 1;
            }
            else
            {
                idx1 = 1;
                idx2 = 1;
            }
        }
        else if (actype == INTER_CAM_CONSTRAINT_FULL || actype == INTER_CAM_CONSTRAINT_PARTIAL)
        {
            if (i==0)
            {
                idx1 = 0;
                idx2 = 1;
            }
            else
            {
                idx1 = 1;
                idx2 = 0;
            }
        }
        else if (actype == INTRA_CAM_CONSTRAINT_FULL || actype == INTRA_CAM_CONSTRAINT_PARTIAL)
        {
            if (i==0)
            {
                idx1 = 0;
                idx2 = 0;
            }
            else
            {
                idx1 = 1;
                idx2 = 1;
            }
        }
        else
        {
            return;
        }
        
        Eigen::Matrix3d R1 = R_camera[idx1];
        Eigen::Vector3d T1 = T_camera[idx1];
        Eigen::Matrix3d R2 = R_camera[idx2];
        Eigen::Vector3d T2 = T_camera[idx2];
        Eigen::Matrix3d Atemp = Ac[i].transpose()*R2.transpose();
        
        Eigen::Matrix<double,6,1> Line_i, Line_j;
        Eigen::Vector3d V = R1*U1;
        Line_i.block(0, 0, 3, 1) = V;
        Line_i.block(3, 0, 3, 1) = T1.cross(V);
        V = R2*U2;
        Line_j.block(0, 0, 3, 1) = V;
        Line_j.block(3, 0, 3, 1) = T2.cross(V);

        double L11 = Line_i(0);
        double L12 = Line_i(1);
        double L13 = Line_i(2);
        double L14 = Line_i(3);
        double L15 = Line_i(4);
        double L16 = Line_i(5);

        double L21 = Line_j(0);
        double L22 = Line_j(1);
        double L23 = Line_j(2);
        double L24 = Line_j(3);
        double L25 = Line_j(4);
        double L26 = Line_j(5);

        double tx1 = T1(0);
        double ty1 = T1(1);
        double tz1 = T1(2);

        double tx2 = T2(0);
        double ty2 = T2(1);
        double tz2 = T2(2);

        double r1 = R1(0,0);
        double r2 = R1(0,1);
        double r3 = R1(0,2);
        double r4 = R1(1,0);
        double r5 = R1(1,1);
        double r6 = R1(1,2);
        double r7 = R1(2,0);
        double r8 = R1(2,1);
        double r9 = R1(2,2);

        double a1 = Atemp(0,0);
        double a2 = Atemp(0,1);
        double a3 = Atemp(0,2);
        double a4 = Atemp(1,0);
        double a5 = Atemp(1,1);
        double a6 = Atemp(1,2);

        Eigen::Vector3d P1_rotated = R1*P1;
        double p11 = P1_rotated(0);
        double p12 = P1_rotated(1);
        double p13 = P1_rotated(2);

        Eigen::Vector3d P2_rotated = R2*P2;
        double p21 = P2_rotated(0);
        double p22 = P2_rotated(1);
        double p23 = P2_rotated(2);

        f1_New_C11.block(i, 0, 1, 10) << L13*L22 - L12*L23, 2*L11*L23, -2*L11*L22, - 2*L12*L22 - 2*L13*L23, L12*L23 + L13*L22, 2*L13*L23 - 2*L12*L22, 2*L11*L22, - L12*L23 - L13*L22, 2*L11*L23, L12*L23 - L13*L22;
        f1_New_C12.block(i, 0, 1, 10) << - L11*L23 - L13*L21, -2*L12*L23, 2*L11*L21 - 2*L13*L23, 2*L12*L21, L11*L23 - L13*L21, 2*L12*L21, - 2*L11*L21 - 2*L13*L23, L11*L23 + L13*L21, 2*L12*L23, L13*L21 - L11*L23;
        f1_New_C13.block(i, 0, 1, 10) << L11*L22 + L12*L21, 2*L12*L22 - 2*L11*L21, 2*L13*L22, 2*L13*L21, - L11*L22 - L12*L21, -2*L13*L21, 2*L13*L22, L12*L21 - L11*L22, - 2*L11*L21 - 2*L12*L22, L11*L22 - L12*L21;
        f1_New_C14.block(i, 0, 1, 10) << L11*L24 + L14*L21 - L12*L25 - L15*L22 - L13*L26 - L16*L23, 2*L11*L25 + 2*L12*L24 + 2*L14*L22 + 2*L15*L21, 2*L11*L26 + 2*L13*L24 + 2*L14*L23 + 2*L16*L21, 2*L12*L26 - 2*L13*L25 + 2*L15*L23 - 2*L16*L22, L12*L25 - L14*L21 - L11*L24 + L15*L22 - L13*L26 - L16*L23, 2*L12*L26 + 2*L13*L25 + 2*L15*L23 + 2*L16*L22, 2*L13*L24 - 2*L11*L26 - 2*L14*L23 + 2*L16*L21, L13*L26 - L14*L21 - L12*L25 - L15*L22 - L11*L24 + L16*L23, 2*L11*L25 - 2*L12*L24 + 2*L14*L22 - 2*L15*L21, L11*L24 + L14*L21 + L12*L25 + L15*L22 + L13*L26 + L16*L23;
        f2_New_C21.block(i, 0, 1, 10) << a3*p12 - a2*p13 + p23*r4 - p22*r7, - 2*a3*p11 - 2*p23*r1, 2*a2*p11 + 2*p22*r1, 2*a2*p12 + 2*a3*p13 + 2*p22*r4 + 2*p23*r7, - a2*p13 - a3*p12 - p23*r4 - p22*r7, 2*a2*p12 - 2*a3*p13 + 2*p22*r4 - 2*p23*r7, - 2*a2*p11 - 2*p22*r1, a2*p13 + a3*p12 + p23*r4 + p22*r7, - 2*a3*p11 - 2*p23*r1, a2*p13 - a3*p12 - p23*r4 + p22*r7;
        f2_New_C22.block(i, 0, 1, 10) << a1*p13 + a3*p11 + p23*r1 + p21*r7, 2*a3*p12 + 2*p23*r4, 2*a3*p13 - 2*a1*p11 - 2*p21*r1 + 2*p23*r7, - 2*a1*p12 - 2*p21*r4, a1*p13 - a3*p11 - p23*r1 + p21*r7, - 2*a1*p12 - 2*p21*r4, 2*a1*p11 + 2*a3*p13 + 2*p21*r1 + 2*p23*r7, - a1*p13 - a3*p11 - p23*r1 - p21*r7, - 2*a3*p12 - 2*p23*r4, a3*p11 - a1*p13 + p23*r1 - p21*r7;
        f2_New_C23.block(i, 0, 1, 10) << - a1*p12 - a2*p11 - p22*r1 - p21*r4, 2*a1*p11 - 2*a2*p12 + 2*p21*r1 - 2*p22*r4, - 2*a2*p13 - 2*p22*r7, - 2*a1*p13 - 2*p21*r7, a1*p12 + a2*p11 + p22*r1 + p21*r4, 2*a1*p13 + 2*p21*r7, - 2*a2*p13 - 2*p22*r7, a2*p11 - a1*p12 + p22*r1 - p21*r4, 2*a1*p11 + 2*a2*p12 + 2*p21*r1 + 2*p22*r4, a1*p12 - a2*p11 - p22*r1 + p21*r4;
        f2_New_C24.block(i, 0, 1, 10) << a3*p12*tx1 - a2*p13*tx1 + a2*p13*tx2 - a3*p12*tx2 - a1*p13*ty1 - a3*p11*ty1 - a1*p13*ty2 - a3*p11*ty2 + a1*p12*tz1 + a2*p11*tz1 + a1*p12*tz2 + a2*p11*tz2 + p23*r4*tx1 - p23*r4*tx2 - p22*r7*tx1 + p22*r7*tx2 - p23*r1*ty1 - p23*r1*ty2 - p21*r7*ty1 - p21*r7*ty2 + p22*r1*tz1 + p22*r1*tz2 + p21*r4*tz1 + p21*r4*tz2, 2*a1*p13*tx1 + 2*a3*p11*tx2 - 2*a2*p13*ty1 - 2*a3*p12*ty2 - 2*a1*p11*tz1 - 2*a1*p11*tz2 + 2*a2*p12*tz1 + 2*a2*p12*tz2 + 2*p23*r1*tx2 + 2*p21*r7*tx1 - 2*p23*r4*ty2 - 2*p22*r7*ty1 - 2*p21*r1*tz1 - 2*p21*r1*tz2 + 2*p22*r4*tz1 + 2*p22*r4*tz2, 2*a1*p11*ty1 - 2*a2*p11*tx2 - 2*a1*p12*tx1 + 2*a1*p11*ty2 - 2*a3*p13*ty1 - 2*a3*p13*ty2 + 2*a3*p12*tz1 + 2*a2*p13*tz2 - 2*p22*r1*tx2 - 2*p21*r4*tx1 + 2*p21*r1*ty1 + 2*p21*r1*ty2 - 2*p23*r7*ty1 - 2*p23*r7*ty2 + 2*p23*r4*tz1 + 2*p22*r7*tz2, 2*a2*p12*tx1 - 2*a2*p12*tx2 + 2*a3*p13*tx1 - 2*a3*p13*tx2 - 2*a2*p11*ty1 + 2*a1*p12*ty2 - 2*a3*p11*tz1 + 2*a1*p13*tz2 + 2*p22*r4*tx1 - 2*p22*r4*tx2 + 2*p23*r7*tx1 - 2*p23*r7*tx2 - 2*p22*r1*ty1 + 2*p21*r4*ty2 - 2*p23*r1*tz1 + 2*p21*r7*tz2, a2*p13*tx1 + a3*p12*tx1 + a2*p13*tx2 + a3*p12*tx2 + a1*p13*ty1 - a3*p11*ty1 - a1*p13*ty2 + a3*p11*ty2 - a1*p12*tz1 - a2*p11*tz1 - a1*p12*tz2 - a2*p11*tz2 + p23*r4*tx1 + p23*r4*tx2 + p22*r7*tx1 + p22*r7*tx2 - p23*r1*ty1 + p23*r1*ty2 + p21*r7*ty1 - p21*r7*ty2 - p22*r1*tz1 - p22*r1*tz2 - p21*r4*tz1 - p21*r4*tz2, 2*a3*p13*tx1 - 2*a2*p12*tx2 - 2*a2*p12*tx1 + 2*a3*p13*tx2 + 2*a2*p11*ty1 + 2*a1*p12*ty2 - 2*a3*p11*tz1 - 2*a1*p13*tz2 - 2*p22*r4*tx1 - 2*p22*r4*tx2 + 2*p23*r7*tx1 + 2*p23*r7*tx2 + 2*p22*r1*ty1 + 2*p21*r4*ty2 - 2*p23*r1*tz1 - 2*p21*r7*tz2, 2*a2*p11*tx2 - 2*a1*p12*tx1 + 2*a1*p11*ty1 - 2*a1*p11*ty2 + 2*a3*p13*ty1 - 2*a3*p13*ty2 - 2*a3*p12*tz1 + 2*a2*p13*tz2 + 2*p22*r1*tx2 - 2*p21*r4*tx1 + 2*p21*r1*ty1 - 2*p21*r1*ty2 + 2*p23*r7*ty1 - 2*p23*r7*ty2 - 2*p23*r4*tz1 + 2*p22*r7*tz2, a1*p13*ty1 - a3*p12*tx1 - a2*p13*tx2 - a3*p12*tx2 - a2*p13*tx1 + a3*p11*ty1 + a1*p13*ty2 + a3*p11*ty2 - a1*p12*tz1 + a2*p11*tz1 + a1*p12*tz2 - a2*p11*tz2 - p23*r4*tx1 - p23*r4*tx2 - p22*r7*tx1 - p22*r7*tx2 + p23*r1*ty1 + p23*r1*ty2 + p21*r7*ty1 + p21*r7*ty2 + p22*r1*tz1 - p22*r1*tz2 - p21*r4*tz1 + p21*r4*tz2, 2*a3*p11*tx2 - 2*a1*p13*tx1 - 2*a2*p13*ty1 + 2*a3*p12*ty2 + 2*a1*p11*tz1 - 2*a1*p11*tz2 + 2*a2*p12*tz1 - 2*a2*p12*tz2 + 2*p23*r1*tx2 - 2*p21*r7*tx1 + 2*p23*r4*ty2 - 2*p22*r7*ty1 + 2*p21*r1*tz1 - 2*p21*r1*tz2 + 2*p22*r4*tz1 - 2*p22*r4*tz2, a2*p13*tx1 - a3*p12*tx1 - a2*p13*tx2 + a3*p12*tx2 - a1*p13*ty1 + a3*p11*ty1 + a1*p13*ty2 - a3*p11*ty2 + a1*p12*tz1 - a2*p11*tz1 - a1*p12*tz2 + a2*p11*tz2 - p23*r4*tx1 + p23*r4*tx2 + p22*r7*tx1 - p22*r7*tx2 + p23*r1*ty1 - p23*r1*ty2 - p21*r7*ty1 + p21*r7*ty2 - p22*r1*tz1 + p22*r1*tz2 + p21*r4*tz1 - p21*r4*tz2;
        f3_New_C31.block(i, 0, 1, 10) << a6*p12 - a5*p13 + p23*r5 - p22*r8, - 2*a6*p11 - 2*p23*r2, 2*a5*p11 + 2*p22*r2, 2*a5*p12 + 2*a6*p13 + 2*p22*r5 + 2*p23*r8, - a5*p13 - a6*p12 - p23*r5 - p22*r8, 2*a5*p12 - 2*a6*p13 + 2*p22*r5 - 2*p23*r8, - 2*a5*p11 - 2*p22*r2, a5*p13 + a6*p12 + p23*r5 + p22*r8, - 2*a6*p11 - 2*p23*r2, a5*p13 - a6*p12 - p23*r5 + p22*r8;
        f3_New_C32.block(i, 0, 1, 10) << a4*p13 + a6*p11 + p23*r2 + p21*r8, 2*a6*p12 + 2*p23*r5, 2*a6*p13 - 2*a4*p11 - 2*p21*r2 + 2*p23*r8, - 2*a4*p12 - 2*p21*r5, a4*p13 - a6*p11 - p23*r2 + p21*r8, - 2*a4*p12 - 2*p21*r5, 2*a4*p11 + 2*a6*p13 + 2*p21*r2 + 2*p23*r8, - a4*p13 - a6*p11 - p23*r2 - p21*r8, - 2*a6*p12 - 2*p23*r5, a6*p11 - a4*p13 + p23*r2 - p21*r8;
        f3_New_C33.block(i, 0, 1, 10) << - a4*p12 - a5*p11 - p22*r2 - p21*r5, 2*a4*p11 - 2*a5*p12 + 2*p21*r2 - 2*p22*r5, - 2*a5*p13 - 2*p22*r8, - 2*a4*p13 - 2*p21*r8, a4*p12 + a5*p11 + p22*r2 + p21*r5, 2*a4*p13 + 2*p21*r8, - 2*a5*p13 - 2*p22*r8, a5*p11 - a4*p12 + p22*r2 - p21*r5, 2*a4*p11 + 2*a5*p12 + 2*p21*r2 + 2*p22*r5, a4*p12 - a5*p11 - p22*r2 + p21*r5;
        f3_New_C34.block(i, 0, 1, 10) << a6*p12*tx1 - a5*p13*tx1 + a5*p13*tx2 - a6*p12*tx2 - a4*p13*ty1 - a6*p11*ty1 - a4*p13*ty2 - a6*p11*ty2 + a4*p12*tz1 + a5*p11*tz1 + a4*p12*tz2 + a5*p11*tz2 + p23*r5*tx1 - p23*r5*tx2 - p22*r8*tx1 + p22*r8*tx2 - p23*r2*ty1 - p23*r2*ty2 - p21*r8*ty1 - p21*r8*ty2 + p22*r2*tz1 + p22*r2*tz2 + p21*r5*tz1 + p21*r5*tz2, 2*a4*p13*tx1 + 2*a6*p11*tx2 - 2*a5*p13*ty1 - 2*a6*p12*ty2 - 2*a4*p11*tz1 - 2*a4*p11*tz2 + 2*a5*p12*tz1 + 2*a5*p12*tz2 + 2*p23*r2*tx2 + 2*p21*r8*tx1 - 2*p23*r5*ty2 - 2*p22*r8*ty1 - 2*p21*r2*tz1 - 2*p21*r2*tz2 + 2*p22*r5*tz1 + 2*p22*r5*tz2, 2*a4*p11*ty1 - 2*a5*p11*tx2 - 2*a4*p12*tx1 + 2*a4*p11*ty2 - 2*a6*p13*ty1 - 2*a6*p13*ty2 + 2*a6*p12*tz1 + 2*a5*p13*tz2 - 2*p22*r2*tx2 - 2*p21*r5*tx1 + 2*p21*r2*ty1 + 2*p21*r2*ty2 - 2*p23*r8*ty1 - 2*p23*r8*ty2 + 2*p23*r5*tz1 + 2*p22*r8*tz2, 2*a5*p12*tx1 - 2*a5*p12*tx2 + 2*a6*p13*tx1 - 2*a6*p13*tx2 - 2*a5*p11*ty1 + 2*a4*p12*ty2 - 2*a6*p11*tz1 + 2*a4*p13*tz2 + 2*p22*r5*tx1 - 2*p22*r5*tx2 + 2*p23*r8*tx1 - 2*p23*r8*tx2 - 2*p22*r2*ty1 + 2*p21*r5*ty2 - 2*p23*r2*tz1 + 2*p21*r8*tz2, a5*p13*tx1 + a6*p12*tx1 + a5*p13*tx2 + a6*p12*tx2 + a4*p13*ty1 - a6*p11*ty1 - a4*p13*ty2 + a6*p11*ty2 - a4*p12*tz1 - a5*p11*tz1 - a4*p12*tz2 - a5*p11*tz2 + p23*r5*tx1 + p23*r5*tx2 + p22*r8*tx1 + p22*r8*tx2 - p23*r2*ty1 + p23*r2*ty2 + p21*r8*ty1 - p21*r8*ty2 - p22*r2*tz1 - p22*r2*tz2 - p21*r5*tz1 - p21*r5*tz2, 2*a6*p13*tx1 - 2*a5*p12*tx2 - 2*a5*p12*tx1 + 2*a6*p13*tx2 + 2*a5*p11*ty1 + 2*a4*p12*ty2 - 2*a6*p11*tz1 - 2*a4*p13*tz2 - 2*p22*r5*tx1 - 2*p22*r5*tx2 + 2*p23*r8*tx1 + 2*p23*r8*tx2 + 2*p22*r2*ty1 + 2*p21*r5*ty2 - 2*p23*r2*tz1 - 2*p21*r8*tz2, 2*a5*p11*tx2 - 2*a4*p12*tx1 + 2*a4*p11*ty1 - 2*a4*p11*ty2 + 2*a6*p13*ty1 - 2*a6*p13*ty2 - 2*a6*p12*tz1 + 2*a5*p13*tz2 + 2*p22*r2*tx2 - 2*p21*r5*tx1 + 2*p21*r2*ty1 - 2*p21*r2*ty2 + 2*p23*r8*ty1 - 2*p23*r8*ty2 - 2*p23*r5*tz1 + 2*p22*r8*tz2, a4*p13*ty1 - a6*p12*tx1 - a5*p13*tx2 - a6*p12*tx2 - a5*p13*tx1 + a6*p11*ty1 + a4*p13*ty2 + a6*p11*ty2 - a4*p12*tz1 + a5*p11*tz1 + a4*p12*tz2 - a5*p11*tz2 - p23*r5*tx1 - p23*r5*tx2 - p22*r8*tx1 - p22*r8*tx2 + p23*r2*ty1 + p23*r2*ty2 + p21*r8*ty1 + p21*r8*ty2 + p22*r2*tz1 - p22*r2*tz2 - p21*r5*tz1 + p21*r5*tz2, 2*a6*p11*tx2 - 2*a4*p13*tx1 - 2*a5*p13*ty1 + 2*a6*p12*ty2 + 2*a4*p11*tz1 - 2*a4*p11*tz2 + 2*a5*p12*tz1 - 2*a5*p12*tz2 + 2*p23*r2*tx2 - 2*p21*r8*tx1 + 2*p23*r5*ty2 - 2*p22*r8*ty1 + 2*p21*r2*tz1 - 2*p21*r2*tz2 + 2*p22*r5*tz1 - 2*p22*r5*tz2, a5*p13*tx1 - a6*p12*tx1 - a5*p13*tx2 + a6*p12*tx2 - a4*p13*ty1 + a6*p11*ty1 + a4*p13*ty2 - a6*p11*ty2 + a4*p12*tz1 - a5*p11*tz1 - a4*p12*tz2 + a5*p11*tz2 - p23*r5*tx1 + p23*r5*tx2 + p22*r8*tx1 - p22*r8*tx2 + p23*r2*ty1 - p23*r2*ty2 - p21*r8*ty1 + p21*r8*ty2 - p22*r2*tz1 + p22*r2*tz2 + p21*r5*tz1 - p21*r5*tz2;
    }

    // row 1
    M[0][0] = (f1_New_C11.row(0));
    M[0][1] = (f1_New_C12.row(0));
    M[0][2] = (f1_New_C13.row(0));
    M[0][3] = (f1_New_C14.row(0));
    // row 2
    M[1][0] = (f2_New_C21.row(0));
    M[1][1] = (f2_New_C22.row(0));
    M[1][2] = (f2_New_C23.row(0));
    M[1][3] = (f2_New_C24.row(0));
    // row 3
    M[2][0] = (f3_New_C31.row(0));
    M[2][1] = (f3_New_C32.row(0));
    M[2][2] = (f3_New_C33.row(0));
    M[2][3] = (f3_New_C34.row(0));
    // row 4
    M[3][0] = (f1_New_C11.row(1));
    M[3][1] = (f1_New_C12.row(1));
    M[3][2] = (f1_New_C13.row(1));
    M[3][3] = (f1_New_C14.row(1));
    // row 5
    M[4][0] = (f2_New_C21.row(1));
    M[4][1] = (f2_New_C22.row(1));
    M[4][2] = (f2_New_C23.row(1));
    M[4][3] = (f2_New_C24.row(1));
    // row 6
    M[5][0] = (f3_New_C31.row(1));
    M[5][1] = (f3_New_C32.row(1));
    M[5][2] = (f3_New_C33.row(1));
    M[5][3] = (f3_New_C34.row(1));
    return;
}

__device__ void create_coeffs(double* coeffs, double* input, Eigen::Matrix<double,1,10>** M,
    double* input_Image_1, double* input_Image_2, double* input_affine_tran,
    double* extrinsic_R_camera, double* extrinsic_T_camera, AC_TYPE actype, 
    bool is_known_angle) {

    Eigen::Matrix3d* R_camera, *Ac;
    Eigen::Vector3d* T_camera, *Image1, *Image2;
    if (actype == GENERIC_CONSTRAINT_FULL || actype == GENERIC_CONSTRAINT_PARTIAL) {
        R_camera = (Eigen::Matrix3d*)malloc(4 * sizeof(Eigen::Matrix3d));
        T_camera = (Eigen::Vector3d*)malloc(4 * sizeof(Eigen::Vector3d));
        Ac = (Eigen::Matrix3d*)malloc(2 * sizeof(Eigen::Matrix3d));
        Image1 = (Eigen::Vector3d*)malloc(2 * sizeof(Eigen::Vector3d));
        Image2 = (Eigen::Vector3d*)malloc(2 * sizeof(Eigen::Vector3d));
        format_convert_generic(input_Image_1, input_Image_2, input_affine_tran, extrinsic_R_camera, extrinsic_T_camera, 
            Image1, Image2, Ac, R_camera, T_camera);
    } else {
        R_camera = (Eigen::Matrix3d*)malloc(2 * sizeof(Eigen::Matrix3d));
        T_camera = (Eigen::Vector3d*)malloc(2 * sizeof(Eigen::Vector3d));
        Ac = (Eigen::Matrix3d*)malloc(2 * sizeof(Eigen::Matrix3d));
        Image1 = (Eigen::Vector3d*)malloc(2 * sizeof(Eigen::Vector3d));
        Image2 = (Eigen::Vector3d*)malloc(2 * sizeof(Eigen::Vector3d));
        format_convert(input_Image_1, input_Image_2, input_affine_tran, extrinsic_R_camera, extrinsic_T_camera, 
            Image1, Image2, Ac, R_camera, T_camera);
    }

    construct_M(M, R_camera, T_camera, Ac, Image1, Image2, actype, is_known_angle);

    free(R_camera);
    free(Ac);
    free(T_camera);
    free(Image1);
    free(Image2);

    Eigen::MatrixXi Sub_index, Sub_index_extra;
    if (!is_known_angle) {
        Sub_index.resize(1,15);
        Sub_index << 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14;

        Sub_index_extra.resize(1,2);
        Sub_index_extra << 0, 1;
    } else {
        Sub_index.resize(1,5);
        Sub_index << 0, 1, 3, 6, 10;

        Sub_index_extra.resize(1,1);
        Sub_index_extra << 0;
    }

    Eigen::Matrix<double,15,84> C;
    Eigen::Matrix<double,15,165> C8;
    construct_order6_poly(M, C, C8, Sub_index);

    Eigen::Matrix<double,2,84> C_extra;
    Eigen::Matrix<double,2,165> C8_extra;
    if (actype == GENERIC_CONSTRAINT_FULL || actype == CASE5_CONSTRAINT_FULL 
        || actype == INTER_CAM_CONSTRAINT_FULL || actype == INTRA_CAM_CONSTRAINT_FULL) {
        construct_order6_extra_poly(M, C_extra, C8_extra, Sub_index_extra);
    }

    // prepare data for Matlab interface
    // Matlab memory is column-major order
    int cnt = 0;
    if (actype == GENERIC_CONSTRAINT_PARTIAL || actype == CASE5_CONSTRAINT_PARTIAL 
        || actype == INTER_CAM_CONSTRAINT_PARTIAL || actype == INTRA_CAM_CONSTRAINT_PARTIAL) {
        for (int j = 0; j < 84; j++) {
            for (int i = 0; i < Sub_index.cols(); i++) {
                coeffs[cnt] = C(Sub_index(i), j);
                cnt++;
            }
        }
    }

    cnt = 0;
    if (actype == GENERIC_CONSTRAINT_FULL || actype == CASE5_CONSTRAINT_FULL 
        || actype == INTER_CAM_CONSTRAINT_FULL || actype == INTRA_CAM_CONSTRAINT_FULL) {
        for (int j = 0; j < 84; j++) {
            for (int i = 0; i < Sub_index.cols(); i++) {
                coeffs[cnt] = C(Sub_index(i), j);
                cnt++;
            }

            for (int i = 0; i < Sub_index_extra.cols(); i++) {
                coeffs[cnt] = C_extra(Sub_index_extra(i), j);
                cnt++;
            }
        }
    }

    // prepare data for the solver
    cnt = 0;
    if (actype == GENERIC_CONSTRAINT_PARTIAL || actype == CASE5_CONSTRAINT_PARTIAL
        ||actype == INTER_CAM_CONSTRAINT_PARTIAL) {
        for (int i = 0; i < Sub_index.cols(); i++) {
            for (int j = 0; j < 84; j++) {
                input[cnt] = C(Sub_index(i), j);
                cnt++;
            }
        }
    }

    cnt = 0;
    if (actype == INTRA_CAM_CONSTRAINT_PARTIAL) {
        for (int i = 0; i < Sub_index.cols(); i++) {
            for (int j = 0; j < 83; j++) {
                input[cnt] = C(Sub_index(i), j);
                cnt++;
            }
        }
    }

    cnt = 0;
    if (actype == GENERIC_CONSTRAINT_FULL || actype == CASE5_CONSTRAINT_FULL
        || actype == INTER_CAM_CONSTRAINT_FULL) {
        for (int i = 0; i < Sub_index.cols(); i++) {
            for (int j = 0; j < 84; j++) {
                input[cnt] = C(Sub_index(i), j);
                cnt++;
            }
        }

        if (is_known_angle) {
            cnt+=4;
        }

        for (int i = 0; i < Sub_index_extra.cols(); i++) {
            for (int j = 49; j < 84; j++) {
                input[cnt] = C_extra(Sub_index_extra(i), j);
                cnt++;
            }
        }
    }

    cnt = 0;
    if (actype == INTRA_CAM_CONSTRAINT_FULL) {
        for (int i = 0; i < Sub_index.cols(); i++) {
            for (int j = 0; j < 83; j++) {
                input[cnt] = C(Sub_index(i), j);
                cnt++;
            }
        }

        if (is_known_angle) {
            cnt+=4;
        }

        for (int i = 0; i < Sub_index_extra.cols(); i++) {
            for (int j = 49; j < 84; j++) {
                input[cnt] = C_extra(Sub_index_extra(i), j);
                cnt++;
            }
        }
    }
    return;
}

__device__ void calculate_M_6_by_4(Eigen::Matrix<double, 6, 4>& M_double, Eigen::Matrix<double,1,10>** M, double x, double y, double z)
{    
    Eigen::Matrix<double, 10, 1> m;
    m(0) = x*x;
    m(1) = x*y;
    m(2) = x*z;
    m(3) = x;
    m(4) = y*y;
    m(5) = y*z;
    m(6) = y;
    m(7) = z*z;
    m(8) = z;
    m(9) = 1;

    for (int i = 0; i < 6; i++)
    {
        for (int j = 0; j < 4; j++)
        {
            Eigen::Matrix<double,1,1> rslt= M[i][j]*m;
            M_double(i,j) = rslt(0);
        }
    }

    return;
}

__device__ void calculate_translation(
    Eigen::MatrixXd sols, Eigen::Matrix<double,1,10>** M, 
    Eigen::Matrix<double,3,1>* q_arr, Eigen::Matrix<double,3,1>* t_arr, bool is_known_angle) {
    for (int s = 0; s < sols.cols(); s++) {
        double x, y, z;
        x = sols(0, s);
        y = sols(1, s);
        z = sols(2, s);
        
        Eigen::Vector3d q;
        q << x, y, z;

        q_arr[s] = q;

        Eigen::Matrix<double, 6, 4> M_double;
        calculate_M_6_by_4(M_double, M, x, y, z);

        Eigen::Matrix<double, 3, 1> C12;
        if (is_known_angle) {
            Eigen::Matrix<double, 5, 3> C0 = M_double.block(0, 0, 5, 3);
            Eigen::Matrix<double, 5, 1> C1 = M_double.block(0, 3, 5, 1);
            // C12 = -C0.colPivHouseholderQr().solve(C1);
            double** A;
            double** B;
            double** X;
            malloc_matrix(5, 3, &A);
            malloc_matrix(5, 1, &B);
            malloc_matrix(3, 1, &X);

            for (int i = 0; i < 5; i++) 
                for (int j = 0; j < 3; j++) 
                    A[i][j] = C0(i, j);

            for (int i = 0; i < 5; i++) 
                for (int j = 0; j < 1; j++) 
                    B[i][j] = C1(i, j);

            qr::solve(5, 3, 1, A, B, X);
            for (int i = 0; i < 3; i++) 
                for (int j = 0; j < 1; j++) 
                    C12(i, j) = -X[i][j];
            
            free_matrix(5, 3, A);
            free_matrix(5, 1, B);
            free_matrix(3, 1, X);
        } else {
            Eigen::Matrix<double, 6, 3> C0 = M_double.block(0, 0, 6, 3);
            Eigen::Matrix<double, 6, 1> C1 = M_double.block(0, 3, 6, 1);
            // C12 = -C0.colPivHouseholderQr().solve(C1);
            double** A;
            double** B;
            double** X;
            malloc_matrix(6, 3, &A);
            malloc_matrix(6, 1, &B);
            malloc_matrix(3, 1, &X);

            for (int i = 0; i < 6; i++) 
                for (int j = 0; j < 3; j++) 
                    A[i][j] = C0(i, j);

            for (int i = 0; i < 6; i++) 
                for (int j = 0; j < 1; j++) 
                    B[i][j] = C1(i, j);

            qr::solve(6, 3, 1, A, B, X);
            for (int i = 0; i < 3; i++) 
                for (int j = 0; j < 1; j++) 
                    C12(i, j) = -X[i][j];
            
            free_matrix(6, 3, A);
            free_matrix(6, 1, B);
            free_matrix(3, 1, X);
        }
        
        t_arr[s] = C12;
    }

    return;
}