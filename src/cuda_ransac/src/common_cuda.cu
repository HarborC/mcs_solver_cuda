#include "hip/hip_runtime.h"
#include "common_cuda.h"
#include <stdio.h> 
#include "qr_cuda.h"

__device__ int find_first_nonzero(double a[165]) {
    int idx = -1;
    for (int i = 0; i < 165; i++) {
        if (abs(a[i]) > NEAR_ZERO_THRESHOLD) {
            idx = i;
            break;
        }
    }
    return idx;
}

__device__ int find_column495(double a[495], double b[3]) {
    int idx = -1;
    for (int i = 0; i < 165; i++) {
        if (a[0*165+i]==b[0] && a[1*165+i]==b[1] && a[2*165+i]==b[2]) {
            idx = i;
            break;
        }
    }
    return idx;
}

__device__ int find_column252(double a[252], double b[3]) {
    int idx = -1;
    for (int i = 0; i < 84; i++) {
        if (a[0*84+i]==b[0] && a[1*84+i]==b[1] && a[2*84+i]==b[2]) { 
            idx = i;
            break;
        }
    }
    return idx;
}

__device__ void quot_var3_order8_by_x2y2z2_1(double c0[165], double c_quot[84]) {
    double c1[165];
    for (int i = 0; i < 165; i++) {
        c1[i] = c0[i];
    }

    for (int i = 0; i < 84; i++) {
        c_quot[i] = 0;
    }

    double M1[495] = { 8, 7, 7, 7, 6, 6, 6, 6, 6, 6, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
                       0, 1, 0, 0, 2, 1, 1, 0, 0, 0, 3, 2, 2, 1, 1, 1, 0, 0, 0, 0, 4, 3, 3, 2, 2, 2, 1, 1, 1, 1, 0, 0, 0, 0, 0, 5, 4, 4, 3, 3, 3, 2, 2, 2, 2, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 6, 5, 5, 4, 4, 4, 3, 3, 3, 3, 2, 2, 2, 2, 2, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 7, 6, 6, 5, 5, 5, 4, 4, 4, 4, 3, 3, 3, 3, 3, 2, 2, 2, 2, 2, 2, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 8, 7, 7, 6, 6, 6, 5, 5, 5, 5, 4, 4, 4, 4, 4, 3, 3, 3, 3, 3, 3, 2, 2, 2, 2, 2, 2, 2, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
                       0, 0, 1, 0, 0, 1, 0, 2, 1, 0, 0, 1, 0, 2, 1, 0, 3, 2, 1, 0, 0, 1, 0, 2, 1, 0, 3, 2, 1, 0, 4, 3, 2, 1, 0, 0, 1, 0, 2, 1, 0, 3, 2, 1, 0, 4, 3, 2, 1, 0, 5, 4, 3, 2, 1, 0, 0, 1, 0, 2, 1, 0, 3, 2, 1, 0, 4, 3, 2, 1, 0, 5, 4, 3, 2, 1, 0, 6, 5, 4, 3, 2, 1, 0, 0, 1, 0, 2, 1, 0, 3, 2, 1, 0, 4, 3, 2, 1, 0, 5, 4, 3, 2, 1, 0, 6, 5, 4, 3, 2, 1, 0, 7, 6, 5, 4, 3, 2, 1, 0, 0, 1, 0, 2, 1, 0, 3, 2, 1, 0, 4, 3, 2, 1, 0, 5, 4, 3, 2, 1, 0, 6, 5, 4, 3, 2, 1, 0, 7, 6, 5, 4, 3, 2, 1, 0, 8, 7, 6, 5, 4, 3, 2, 1, 0 };
    double M2[12] = { 2, 0, 0, 0, 
                      0, 2, 0, 0,
                      0, 0, 2, 0 };
    double M_quot[252] = { 6, 5, 4, 3, 2, 1, 0, 5, 4, 3, 2, 1, 0, 4, 3, 2, 1, 0, 3, 2, 1, 0, 2, 1, 0, 1, 0, 0, 5, 4, 3, 2, 1, 0, 4, 3, 2, 1, 0, 3, 2, 1, 0, 2, 1, 0, 1, 0, 0, 4, 3, 2, 1, 0, 3, 2, 1, 0, 2, 1, 0, 1, 0, 0, 3, 2, 1, 0, 2, 1, 0, 1, 0, 0, 2, 1, 0, 1, 0, 0, 1, 0, 0, 0, 
                           0, 1, 2, 3, 4, 5, 6, 0, 1, 2, 3, 4, 5, 0, 1, 2, 3, 4, 0, 1, 2, 3, 0, 1, 2, 0, 1, 0, 0, 1, 2, 3, 4, 5, 0, 1, 2, 3, 4, 0, 1, 2, 3, 0, 1, 2, 0, 1, 0, 0, 1, 2, 3, 4, 0, 1, 2, 3, 0, 1, 2, 0, 1, 0, 0, 1, 2, 3, 0, 1, 2, 0, 1, 0, 0, 1, 2, 0, 1, 0, 0, 1, 0, 0, 
                           0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 2, 2, 2, 2, 2, 3, 3, 3, 3, 4, 4, 4, 5, 5, 6, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 2, 2, 2, 2, 3, 3, 3, 4, 4, 5, 0, 0, 0, 0, 0, 1, 1, 1, 1, 2, 2, 2, 3, 3, 4, 0, 0, 0, 0, 1, 1, 1, 2, 2, 3, 0, 0, 0, 1, 1, 2, 0, 0, 1, 0 };
    
    double c2[4] = {1, 1, 1, 1};
    
    double LMid[3], m[3], LMd[3];
    for (int j = 0; j < 3; j++) {
        LMd[j] = M2[j*4+0];
    }

    while(true) {
        int idx = find_first_nonzero(c1);
        if (idx==-1)
            break;

        for (int j = 0; j < 3; j++) {
            LMid[j] = M1[j*165 + idx];
        }
        bool divisible = false;

        for (int j = 0; j < 3; j++) {
            m[j] = LMid[j] - LMd[j];
        }

        if (m[0]>=0 && m[1]>=0 && m[2]>=0) {
            double c = c1[idx]/c2[0];
            int idx2 = find_column252(M_quot, m);
            if (idx2==-1) {
                continue;
            }

            c_quot[idx2] = c_quot[idx2] + c;
            
            for (int k = 0; k < 4; k++) {
                double m_tmp[3];
                for (int j = 0; j < 3; j++) {
                    m_tmp[j] = m[j] + M2[j*4+k];
                }
                
                double c_tmp = c*c2[k];
                int idx3 = find_column495(M1, m_tmp);

                if (idx3==-1) {
                    continue;
                }
                
                c1[idx3] =  c1[idx3] - c_tmp;
            }
            divisible = true;
        }

        if (!divisible) {
            int idx = find_first_nonzero(c1);
            if (idx==-1)
                break;
            c1[idx] = 0;
        }
    }
}

__device__ void construct_order6_poly(Eigen::Matrix<double,1,10> M[6][4], double C[1260], int *Sub_index, int N) {
    for (int i = 0; i < 1260; i++) {
        C[i] = 0;
    }

    int Idx_all[60] = { 0, 1, 2, 3, 
                          0, 1, 2, 4, 
                          0, 1, 2, 5,
                          0, 1, 3, 4,
                          0, 1, 3, 5,
                          0, 1, 4, 5,
                          0, 2, 3, 4,
                          0, 2, 3, 5,
                          0, 2, 4, 5,
                          0, 3, 4, 5,
                          1, 2, 3, 4,
                          1, 2, 3, 5,
                          1, 2, 4, 5,
                          1, 3, 4, 5,
                          2, 3, 4, 5};
    
    for (int k = 0; k < N; k++) {
        int i = Sub_index[k];
        double idx[4];
        for (int j = 0; j < 4; j++) {
            idx[j] = Idx_all[i*4+j];
        }

        double m11[10], m12[10], m13[10], m14[10], 
               m21[10], m22[10], m23[10], m24[10],
               m31[10], m32[10], m33[10], m34[10],
               m41[10], m42[10], m43[10], m44[10];

        for (int j = 0; j < 10; j++) {
            int idx0 = idx[0], idx1 = idx[1], idx2 = idx[2], idx3 = idx[3];
            m11[j] = M[idx0][0](j); m12[j] = M[idx0][1](j); m13[j] = M[idx0][2](j), m14[j] = M[idx0][3](j);
            m21[j] = M[idx1][0](j); m22[j] = M[idx1][1](j); m23[j] = M[idx1][2](j), m24[j] = M[idx1][3](j);
            m31[j] = M[idx2][0](j); m32[j] = M[idx2][1](j); m33[j] = M[idx2][2](j), m34[j] = M[idx2][3](j);
            m41[j] = M[idx3][0](j); m42[j] = M[idx3][1](j); m43[j] = M[idx3][2](j), m44[j] = M[idx3][3](j);
        }

        double d[165], d0[165];
        var3_order2_four_multiplication(m11, m22, m33, m44, d0); 
        for (int j = 0; j < 165; j++) {
            d[j] = d0[j];
        }
        var3_order2_four_multiplication(m11, m22, m34, m43, d0); 
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] - d0[j];
        }
        var3_order2_four_multiplication(m11, m23, m32, m44, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] - d0[j];
        }
        var3_order2_four_multiplication(m11, m23, m34, m42, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] + d0[j];
        }
        var3_order2_four_multiplication(m11, m24, m32, m43, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] + d0[j];
        }
        var3_order2_four_multiplication(m11, m24, m33, m42, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] - d0[j];
        }
        var3_order2_four_multiplication(m12, m21, m33, m44, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] - d0[j];
        }
        var3_order2_four_multiplication(m12, m21, m34, m43, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] + d0[j];
        }
        var3_order2_four_multiplication(m12, m23, m31, m44, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] + d0[j];
        }
        var3_order2_four_multiplication(m12, m23, m34, m41, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] - d0[j];
        }
        var3_order2_four_multiplication(m12, m24, m31, m43, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] - d0[j];
        }
        var3_order2_four_multiplication(m12, m24, m33, m41, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] + d0[j];
        }
        var3_order2_four_multiplication(m13, m21, m32, m44, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] + d0[j];
        }
        var3_order2_four_multiplication(m13, m21, m34, m42, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] - d0[j];
        }
        var3_order2_four_multiplication(m13, m22, m31, m44, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] - d0[j];
        }
        var3_order2_four_multiplication(m13, m22, m34, m41, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] + d0[j];
        }
        var3_order2_four_multiplication(m13, m24, m31, m42, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] + d0[j];
        }
        var3_order2_four_multiplication(m13, m24, m32, m41, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] - d0[j];
        }
        var3_order2_four_multiplication(m14, m21, m32, m43, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] - d0[j];
        }
        var3_order2_four_multiplication(m14, m21, m33, m42, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] + d0[j];
        }
        var3_order2_four_multiplication(m14, m22, m31, m43, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] + d0[j];
        }
        var3_order2_four_multiplication(m14, m22, m33, m41, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] - d0[j];
        }
        var3_order2_four_multiplication(m14, m23, m31, m42, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] - d0[j];
        }
        var3_order2_four_multiplication(m14, m23, m32, m41, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] + d0[j];
        }

        double quot[84];
        quot_var3_order8_by_x2y2z2_1(d, quot);
        for (int j = 0; j < 84; j++) {
            C[i*84 + 0 + j] = quot[j];
        }
    }
    return;
}

__device__ void construct_order6_extra_poly(Eigen::Matrix<double,1,10> M[6][4], double C[168], int *Sub_index_extra, int N) {
    for (int i = 0; i < 168; i++) {
        C[i] = 0;
    }

    double Idx_all[6] = { 0, 1, 2, 3, 4, 5 };

    double m_one[10] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 1};

    for (int k = 0; k < N; k++) {
        int i = Sub_index_extra[k];
        int idx[3];
        for (int j = 0; j < 3; j++) {
            idx[j] = Idx_all[i*3+j];
        }

        double m11[10], m12[10], m13[10], 
               m21[10], m22[10], m23[10],
               m31[10], m32[10], m33[10],
               m44[10];

        for (int j = 0; j < 10; j++) {
            int idx0 = idx[0], idx1 = idx[1], idx2 = idx[2];
            m11[j] = M[idx0][0](j); m12[j] = M[idx0][1](j); m13[j] = M[idx0][2](j);
            m21[j] = M[idx1][0](j); m22[j] = M[idx1][1](j); m23[j] = M[idx1][2](j);
            m31[j] = M[idx2][0](j); m32[j] = M[idx2][1](j); m33[j] = M[idx2][2](j);
            m44[j] = m_one[j];
        }
        
        double d[165], d0[165];
        var3_order2_four_multiplication(m11, m22, m33, m44, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d0[j];
        }
        var3_order2_four_multiplication(m11, m23, m32, m44, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] - d0[j];
        }
        var3_order2_four_multiplication(m12, m21, m33, m44, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] - d0[j];
        }
        var3_order2_four_multiplication(m12, m23, m31, m44, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] + d0[j];
        }
        var3_order2_four_multiplication(m13, m21, m32, m44, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] + d0[j];
        }
        var3_order2_four_multiplication(m13, m22, m31, m44, d0);
        for (int j = 0; j < 165; j++) {
            d[j] = d[j] - d0[j];
        }

        double quot[84];
        quot_var3_order8_by_x2y2z2_1(d, quot);
        for (int j = 0; j < 84; j++) {
            C[i * 84 + 0 + j] = quot[j];
        }
    }
    return;
}

__device__ void construct_M(
    Eigen::Matrix<double,1,10> M[6][4], 
    Eigen::Matrix3d* R_camera, Eigen::Vector3d* T_camera, 
    Eigen::Matrix3d* Ac, Eigen::Vector3d* Image1, Eigen::Vector3d* Image2,
    AC_TYPE actype, bool is_known_angle) {

    int point_num = 2;
    Eigen::Matrix<double,2,10> 
        f1_New_C11, f1_New_C12, f1_New_C13, f1_New_C14,
        f2_New_C21, f2_New_C22, f2_New_C23, f2_New_C24,
        f3_New_C31, f3_New_C32, f3_New_C33, f3_New_C34;
    for (int i = 0; i < point_num; i++)
    {
        Eigen::Vector3d P1 = Image1[i];
        Eigen::Vector3d P2 = Image2[i];
        Eigen::Vector3d U1 = P1;
        U1.normalize();
        Eigen::Vector3d U2 = P2;
        U2.normalize();

        int idx1 = 0;
        int idx2 = 0;
		if(actype == GENERIC_CONSTRAINT_FULL || actype == GENERIC_CONSTRAINT_PARTIAL)
        {
            if (i==0)
            {
                idx1 = 0;
                idx2 = 1;
            }
            else
            {
                idx1 = 2;
                idx2 = 3;
            }
        }
        else if (actype == CASE5_CONSTRAINT_FULL || actype == CASE5_CONSTRAINT_PARTIAL)
        {
            if (i==0)
            {
                idx1 = 0;
                idx2 = 1;
            }
            else
            {
                idx1 = 1;
                idx2 = 1;
            }
        }
        else if (actype == INTER_CAM_CONSTRAINT_FULL || actype == INTER_CAM_CONSTRAINT_PARTIAL)
        {
            if (i==0)
            {
                idx1 = 0;
                idx2 = 1;
            }
            else
            {
                idx1 = 1;
                idx2 = 0;
            }
        }
        else if (actype == INTRA_CAM_CONSTRAINT_FULL || actype == INTRA_CAM_CONSTRAINT_PARTIAL)
        {
            if (i==0)
            {
                idx1 = 0;
                idx2 = 0;
            }
            else
            {
                idx1 = 1;
                idx2 = 1;
            }
        }
        else
        {
            return;
        }
        
        Eigen::Matrix3d R1 = R_camera[idx1];
        Eigen::Vector3d T1 = T_camera[idx1];
        Eigen::Matrix3d R2 = R_camera[idx2];
        Eigen::Vector3d T2 = T_camera[idx2];
        Eigen::Matrix3d Atemp = Ac[i].transpose()*R2.transpose();
        
        Eigen::Matrix<double,6,1> Line_i, Line_j;
        Eigen::Vector3d V = R1*U1;
        Line_i.block(0, 0, 3, 1) = V;
        Line_i.block(3, 0, 3, 1) = T1.cross(V);
        V = R2*U2;
        Line_j.block(0, 0, 3, 1) = V;
        Line_j.block(3, 0, 3, 1) = T2.cross(V);

        double L11 = Line_i(0);
        double L12 = Line_i(1);
        double L13 = Line_i(2);
        double L14 = Line_i(3);
        double L15 = Line_i(4);
        double L16 = Line_i(5);

        double L21 = Line_j(0);
        double L22 = Line_j(1);
        double L23 = Line_j(2);
        double L24 = Line_j(3);
        double L25 = Line_j(4);
        double L26 = Line_j(5);

        double tx1 = T1(0);
        double ty1 = T1(1);
        double tz1 = T1(2);

        double tx2 = T2(0);
        double ty2 = T2(1);
        double tz2 = T2(2);

        double r1 = R1(0,0);
        double r2 = R1(0,1);
        double r3 = R1(0,2);
        double r4 = R1(1,0);
        double r5 = R1(1,1);
        double r6 = R1(1,2);
        double r7 = R1(2,0);
        double r8 = R1(2,1);
        double r9 = R1(2,2);

        double a1 = Atemp(0,0);
        double a2 = Atemp(0,1);
        double a3 = Atemp(0,2);
        double a4 = Atemp(1,0);
        double a5 = Atemp(1,1);
        double a6 = Atemp(1,2);

        Eigen::Vector3d P1_rotated = R1*P1;
        double p11 = P1_rotated(0);
        double p12 = P1_rotated(1);
        double p13 = P1_rotated(2);

        Eigen::Vector3d P2_rotated = R2*P2;
        double p21 = P2_rotated(0);
        double p22 = P2_rotated(1);
        double p23 = P2_rotated(2);

        f1_New_C11.block(i, 0, 1, 10) << L13*L22 - L12*L23, 2*L11*L23, -2*L11*L22, - 2*L12*L22 - 2*L13*L23, L12*L23 + L13*L22, 2*L13*L23 - 2*L12*L22, 2*L11*L22, - L12*L23 - L13*L22, 2*L11*L23, L12*L23 - L13*L22;
        f1_New_C12.block(i, 0, 1, 10) << - L11*L23 - L13*L21, -2*L12*L23, 2*L11*L21 - 2*L13*L23, 2*L12*L21, L11*L23 - L13*L21, 2*L12*L21, - 2*L11*L21 - 2*L13*L23, L11*L23 + L13*L21, 2*L12*L23, L13*L21 - L11*L23;
        f1_New_C13.block(i, 0, 1, 10) << L11*L22 + L12*L21, 2*L12*L22 - 2*L11*L21, 2*L13*L22, 2*L13*L21, - L11*L22 - L12*L21, -2*L13*L21, 2*L13*L22, L12*L21 - L11*L22, - 2*L11*L21 - 2*L12*L22, L11*L22 - L12*L21;
        f1_New_C14.block(i, 0, 1, 10) << L11*L24 + L14*L21 - L12*L25 - L15*L22 - L13*L26 - L16*L23, 2*L11*L25 + 2*L12*L24 + 2*L14*L22 + 2*L15*L21, 2*L11*L26 + 2*L13*L24 + 2*L14*L23 + 2*L16*L21, 2*L12*L26 - 2*L13*L25 + 2*L15*L23 - 2*L16*L22, L12*L25 - L14*L21 - L11*L24 + L15*L22 - L13*L26 - L16*L23, 2*L12*L26 + 2*L13*L25 + 2*L15*L23 + 2*L16*L22, 2*L13*L24 - 2*L11*L26 - 2*L14*L23 + 2*L16*L21, L13*L26 - L14*L21 - L12*L25 - L15*L22 - L11*L24 + L16*L23, 2*L11*L25 - 2*L12*L24 + 2*L14*L22 - 2*L15*L21, L11*L24 + L14*L21 + L12*L25 + L15*L22 + L13*L26 + L16*L23;
        f2_New_C21.block(i, 0, 1, 10) << a3*p12 - a2*p13 + p23*r4 - p22*r7, - 2*a3*p11 - 2*p23*r1, 2*a2*p11 + 2*p22*r1, 2*a2*p12 + 2*a3*p13 + 2*p22*r4 + 2*p23*r7, - a2*p13 - a3*p12 - p23*r4 - p22*r7, 2*a2*p12 - 2*a3*p13 + 2*p22*r4 - 2*p23*r7, - 2*a2*p11 - 2*p22*r1, a2*p13 + a3*p12 + p23*r4 + p22*r7, - 2*a3*p11 - 2*p23*r1, a2*p13 - a3*p12 - p23*r4 + p22*r7;
        f2_New_C22.block(i, 0, 1, 10) << a1*p13 + a3*p11 + p23*r1 + p21*r7, 2*a3*p12 + 2*p23*r4, 2*a3*p13 - 2*a1*p11 - 2*p21*r1 + 2*p23*r7, - 2*a1*p12 - 2*p21*r4, a1*p13 - a3*p11 - p23*r1 + p21*r7, - 2*a1*p12 - 2*p21*r4, 2*a1*p11 + 2*a3*p13 + 2*p21*r1 + 2*p23*r7, - a1*p13 - a3*p11 - p23*r1 - p21*r7, - 2*a3*p12 - 2*p23*r4, a3*p11 - a1*p13 + p23*r1 - p21*r7;
        f2_New_C23.block(i, 0, 1, 10) << - a1*p12 - a2*p11 - p22*r1 - p21*r4, 2*a1*p11 - 2*a2*p12 + 2*p21*r1 - 2*p22*r4, - 2*a2*p13 - 2*p22*r7, - 2*a1*p13 - 2*p21*r7, a1*p12 + a2*p11 + p22*r1 + p21*r4, 2*a1*p13 + 2*p21*r7, - 2*a2*p13 - 2*p22*r7, a2*p11 - a1*p12 + p22*r1 - p21*r4, 2*a1*p11 + 2*a2*p12 + 2*p21*r1 + 2*p22*r4, a1*p12 - a2*p11 - p22*r1 + p21*r4;
        f2_New_C24.block(i, 0, 1, 10) << a3*p12*tx1 - a2*p13*tx1 + a2*p13*tx2 - a3*p12*tx2 - a1*p13*ty1 - a3*p11*ty1 - a1*p13*ty2 - a3*p11*ty2 + a1*p12*tz1 + a2*p11*tz1 + a1*p12*tz2 + a2*p11*tz2 + p23*r4*tx1 - p23*r4*tx2 - p22*r7*tx1 + p22*r7*tx2 - p23*r1*ty1 - p23*r1*ty2 - p21*r7*ty1 - p21*r7*ty2 + p22*r1*tz1 + p22*r1*tz2 + p21*r4*tz1 + p21*r4*tz2, 2*a1*p13*tx1 + 2*a3*p11*tx2 - 2*a2*p13*ty1 - 2*a3*p12*ty2 - 2*a1*p11*tz1 - 2*a1*p11*tz2 + 2*a2*p12*tz1 + 2*a2*p12*tz2 + 2*p23*r1*tx2 + 2*p21*r7*tx1 - 2*p23*r4*ty2 - 2*p22*r7*ty1 - 2*p21*r1*tz1 - 2*p21*r1*tz2 + 2*p22*r4*tz1 + 2*p22*r4*tz2, 2*a1*p11*ty1 - 2*a2*p11*tx2 - 2*a1*p12*tx1 + 2*a1*p11*ty2 - 2*a3*p13*ty1 - 2*a3*p13*ty2 + 2*a3*p12*tz1 + 2*a2*p13*tz2 - 2*p22*r1*tx2 - 2*p21*r4*tx1 + 2*p21*r1*ty1 + 2*p21*r1*ty2 - 2*p23*r7*ty1 - 2*p23*r7*ty2 + 2*p23*r4*tz1 + 2*p22*r7*tz2, 2*a2*p12*tx1 - 2*a2*p12*tx2 + 2*a3*p13*tx1 - 2*a3*p13*tx2 - 2*a2*p11*ty1 + 2*a1*p12*ty2 - 2*a3*p11*tz1 + 2*a1*p13*tz2 + 2*p22*r4*tx1 - 2*p22*r4*tx2 + 2*p23*r7*tx1 - 2*p23*r7*tx2 - 2*p22*r1*ty1 + 2*p21*r4*ty2 - 2*p23*r1*tz1 + 2*p21*r7*tz2, a2*p13*tx1 + a3*p12*tx1 + a2*p13*tx2 + a3*p12*tx2 + a1*p13*ty1 - a3*p11*ty1 - a1*p13*ty2 + a3*p11*ty2 - a1*p12*tz1 - a2*p11*tz1 - a1*p12*tz2 - a2*p11*tz2 + p23*r4*tx1 + p23*r4*tx2 + p22*r7*tx1 + p22*r7*tx2 - p23*r1*ty1 + p23*r1*ty2 + p21*r7*ty1 - p21*r7*ty2 - p22*r1*tz1 - p22*r1*tz2 - p21*r4*tz1 - p21*r4*tz2, 2*a3*p13*tx1 - 2*a2*p12*tx2 - 2*a2*p12*tx1 + 2*a3*p13*tx2 + 2*a2*p11*ty1 + 2*a1*p12*ty2 - 2*a3*p11*tz1 - 2*a1*p13*tz2 - 2*p22*r4*tx1 - 2*p22*r4*tx2 + 2*p23*r7*tx1 + 2*p23*r7*tx2 + 2*p22*r1*ty1 + 2*p21*r4*ty2 - 2*p23*r1*tz1 - 2*p21*r7*tz2, 2*a2*p11*tx2 - 2*a1*p12*tx1 + 2*a1*p11*ty1 - 2*a1*p11*ty2 + 2*a3*p13*ty1 - 2*a3*p13*ty2 - 2*a3*p12*tz1 + 2*a2*p13*tz2 + 2*p22*r1*tx2 - 2*p21*r4*tx1 + 2*p21*r1*ty1 - 2*p21*r1*ty2 + 2*p23*r7*ty1 - 2*p23*r7*ty2 - 2*p23*r4*tz1 + 2*p22*r7*tz2, a1*p13*ty1 - a3*p12*tx1 - a2*p13*tx2 - a3*p12*tx2 - a2*p13*tx1 + a3*p11*ty1 + a1*p13*ty2 + a3*p11*ty2 - a1*p12*tz1 + a2*p11*tz1 + a1*p12*tz2 - a2*p11*tz2 - p23*r4*tx1 - p23*r4*tx2 - p22*r7*tx1 - p22*r7*tx2 + p23*r1*ty1 + p23*r1*ty2 + p21*r7*ty1 + p21*r7*ty2 + p22*r1*tz1 - p22*r1*tz2 - p21*r4*tz1 + p21*r4*tz2, 2*a3*p11*tx2 - 2*a1*p13*tx1 - 2*a2*p13*ty1 + 2*a3*p12*ty2 + 2*a1*p11*tz1 - 2*a1*p11*tz2 + 2*a2*p12*tz1 - 2*a2*p12*tz2 + 2*p23*r1*tx2 - 2*p21*r7*tx1 + 2*p23*r4*ty2 - 2*p22*r7*ty1 + 2*p21*r1*tz1 - 2*p21*r1*tz2 + 2*p22*r4*tz1 - 2*p22*r4*tz2, a2*p13*tx1 - a3*p12*tx1 - a2*p13*tx2 + a3*p12*tx2 - a1*p13*ty1 + a3*p11*ty1 + a1*p13*ty2 - a3*p11*ty2 + a1*p12*tz1 - a2*p11*tz1 - a1*p12*tz2 + a2*p11*tz2 - p23*r4*tx1 + p23*r4*tx2 + p22*r7*tx1 - p22*r7*tx2 + p23*r1*ty1 - p23*r1*ty2 - p21*r7*ty1 + p21*r7*ty2 - p22*r1*tz1 + p22*r1*tz2 + p21*r4*tz1 - p21*r4*tz2;
        f3_New_C31.block(i, 0, 1, 10) << a6*p12 - a5*p13 + p23*r5 - p22*r8, - 2*a6*p11 - 2*p23*r2, 2*a5*p11 + 2*p22*r2, 2*a5*p12 + 2*a6*p13 + 2*p22*r5 + 2*p23*r8, - a5*p13 - a6*p12 - p23*r5 - p22*r8, 2*a5*p12 - 2*a6*p13 + 2*p22*r5 - 2*p23*r8, - 2*a5*p11 - 2*p22*r2, a5*p13 + a6*p12 + p23*r5 + p22*r8, - 2*a6*p11 - 2*p23*r2, a5*p13 - a6*p12 - p23*r5 + p22*r8;
        f3_New_C32.block(i, 0, 1, 10) << a4*p13 + a6*p11 + p23*r2 + p21*r8, 2*a6*p12 + 2*p23*r5, 2*a6*p13 - 2*a4*p11 - 2*p21*r2 + 2*p23*r8, - 2*a4*p12 - 2*p21*r5, a4*p13 - a6*p11 - p23*r2 + p21*r8, - 2*a4*p12 - 2*p21*r5, 2*a4*p11 + 2*a6*p13 + 2*p21*r2 + 2*p23*r8, - a4*p13 - a6*p11 - p23*r2 - p21*r8, - 2*a6*p12 - 2*p23*r5, a6*p11 - a4*p13 + p23*r2 - p21*r8;
        f3_New_C33.block(i, 0, 1, 10) << - a4*p12 - a5*p11 - p22*r2 - p21*r5, 2*a4*p11 - 2*a5*p12 + 2*p21*r2 - 2*p22*r5, - 2*a5*p13 - 2*p22*r8, - 2*a4*p13 - 2*p21*r8, a4*p12 + a5*p11 + p22*r2 + p21*r5, 2*a4*p13 + 2*p21*r8, - 2*a5*p13 - 2*p22*r8, a5*p11 - a4*p12 + p22*r2 - p21*r5, 2*a4*p11 + 2*a5*p12 + 2*p21*r2 + 2*p22*r5, a4*p12 - a5*p11 - p22*r2 + p21*r5;
        f3_New_C34.block(i, 0, 1, 10) << a6*p12*tx1 - a5*p13*tx1 + a5*p13*tx2 - a6*p12*tx2 - a4*p13*ty1 - a6*p11*ty1 - a4*p13*ty2 - a6*p11*ty2 + a4*p12*tz1 + a5*p11*tz1 + a4*p12*tz2 + a5*p11*tz2 + p23*r5*tx1 - p23*r5*tx2 - p22*r8*tx1 + p22*r8*tx2 - p23*r2*ty1 - p23*r2*ty2 - p21*r8*ty1 - p21*r8*ty2 + p22*r2*tz1 + p22*r2*tz2 + p21*r5*tz1 + p21*r5*tz2, 2*a4*p13*tx1 + 2*a6*p11*tx2 - 2*a5*p13*ty1 - 2*a6*p12*ty2 - 2*a4*p11*tz1 - 2*a4*p11*tz2 + 2*a5*p12*tz1 + 2*a5*p12*tz2 + 2*p23*r2*tx2 + 2*p21*r8*tx1 - 2*p23*r5*ty2 - 2*p22*r8*ty1 - 2*p21*r2*tz1 - 2*p21*r2*tz2 + 2*p22*r5*tz1 + 2*p22*r5*tz2, 2*a4*p11*ty1 - 2*a5*p11*tx2 - 2*a4*p12*tx1 + 2*a4*p11*ty2 - 2*a6*p13*ty1 - 2*a6*p13*ty2 + 2*a6*p12*tz1 + 2*a5*p13*tz2 - 2*p22*r2*tx2 - 2*p21*r5*tx1 + 2*p21*r2*ty1 + 2*p21*r2*ty2 - 2*p23*r8*ty1 - 2*p23*r8*ty2 + 2*p23*r5*tz1 + 2*p22*r8*tz2, 2*a5*p12*tx1 - 2*a5*p12*tx2 + 2*a6*p13*tx1 - 2*a6*p13*tx2 - 2*a5*p11*ty1 + 2*a4*p12*ty2 - 2*a6*p11*tz1 + 2*a4*p13*tz2 + 2*p22*r5*tx1 - 2*p22*r5*tx2 + 2*p23*r8*tx1 - 2*p23*r8*tx2 - 2*p22*r2*ty1 + 2*p21*r5*ty2 - 2*p23*r2*tz1 + 2*p21*r8*tz2, a5*p13*tx1 + a6*p12*tx1 + a5*p13*tx2 + a6*p12*tx2 + a4*p13*ty1 - a6*p11*ty1 - a4*p13*ty2 + a6*p11*ty2 - a4*p12*tz1 - a5*p11*tz1 - a4*p12*tz2 - a5*p11*tz2 + p23*r5*tx1 + p23*r5*tx2 + p22*r8*tx1 + p22*r8*tx2 - p23*r2*ty1 + p23*r2*ty2 + p21*r8*ty1 - p21*r8*ty2 - p22*r2*tz1 - p22*r2*tz2 - p21*r5*tz1 - p21*r5*tz2, 2*a6*p13*tx1 - 2*a5*p12*tx2 - 2*a5*p12*tx1 + 2*a6*p13*tx2 + 2*a5*p11*ty1 + 2*a4*p12*ty2 - 2*a6*p11*tz1 - 2*a4*p13*tz2 - 2*p22*r5*tx1 - 2*p22*r5*tx2 + 2*p23*r8*tx1 + 2*p23*r8*tx2 + 2*p22*r2*ty1 + 2*p21*r5*ty2 - 2*p23*r2*tz1 - 2*p21*r8*tz2, 2*a5*p11*tx2 - 2*a4*p12*tx1 + 2*a4*p11*ty1 - 2*a4*p11*ty2 + 2*a6*p13*ty1 - 2*a6*p13*ty2 - 2*a6*p12*tz1 + 2*a5*p13*tz2 + 2*p22*r2*tx2 - 2*p21*r5*tx1 + 2*p21*r2*ty1 - 2*p21*r2*ty2 + 2*p23*r8*ty1 - 2*p23*r8*ty2 - 2*p23*r5*tz1 + 2*p22*r8*tz2, a4*p13*ty1 - a6*p12*tx1 - a5*p13*tx2 - a6*p12*tx2 - a5*p13*tx1 + a6*p11*ty1 + a4*p13*ty2 + a6*p11*ty2 - a4*p12*tz1 + a5*p11*tz1 + a4*p12*tz2 - a5*p11*tz2 - p23*r5*tx1 - p23*r5*tx2 - p22*r8*tx1 - p22*r8*tx2 + p23*r2*ty1 + p23*r2*ty2 + p21*r8*ty1 + p21*r8*ty2 + p22*r2*tz1 - p22*r2*tz2 - p21*r5*tz1 + p21*r5*tz2, 2*a6*p11*tx2 - 2*a4*p13*tx1 - 2*a5*p13*ty1 + 2*a6*p12*ty2 + 2*a4*p11*tz1 - 2*a4*p11*tz2 + 2*a5*p12*tz1 - 2*a5*p12*tz2 + 2*p23*r2*tx2 - 2*p21*r8*tx1 + 2*p23*r5*ty2 - 2*p22*r8*ty1 + 2*p21*r2*tz1 - 2*p21*r2*tz2 + 2*p22*r5*tz1 - 2*p22*r5*tz2, a5*p13*tx1 - a6*p12*tx1 - a5*p13*tx2 + a6*p12*tx2 - a4*p13*ty1 + a6*p11*ty1 + a4*p13*ty2 - a6*p11*ty2 + a4*p12*tz1 - a5*p11*tz1 - a4*p12*tz2 + a5*p11*tz2 - p23*r5*tx1 + p23*r5*tx2 + p22*r8*tx1 - p22*r8*tx2 + p23*r2*ty1 - p23*r2*ty2 - p21*r8*ty1 + p21*r8*ty2 - p22*r2*tz1 + p22*r2*tz2 + p21*r5*tz1 - p21*r5*tz2;
    }

    // row 1
    M[0][0] = (f1_New_C11.row(0));
    M[0][1] = (f1_New_C12.row(0));
    M[0][2] = (f1_New_C13.row(0));
    M[0][3] = (f1_New_C14.row(0));
    // row 2
    M[1][0] = (f2_New_C21.row(0));
    M[1][1] = (f2_New_C22.row(0));
    M[1][2] = (f2_New_C23.row(0));
    M[1][3] = (f2_New_C24.row(0));
    // row 3
    M[2][0] = (f3_New_C31.row(0));
    M[2][1] = (f3_New_C32.row(0));
    M[2][2] = (f3_New_C33.row(0));
    M[2][3] = (f3_New_C34.row(0));
    // row 4
    M[3][0] = (f1_New_C11.row(1));
    M[3][1] = (f1_New_C12.row(1));
    M[3][2] = (f1_New_C13.row(1));
    M[3][3] = (f1_New_C14.row(1));
    // row 5
    M[4][0] = (f2_New_C21.row(1));
    M[4][1] = (f2_New_C22.row(1));
    M[4][2] = (f2_New_C23.row(1));
    M[4][3] = (f2_New_C24.row(1));
    // row 6
    M[5][0] = (f3_New_C31.row(1));
    M[5][1] = (f3_New_C32.row(1));
    M[5][2] = (f3_New_C33.row(1));
    M[5][3] = (f3_New_C34.row(1));
    return;
}

__device__ void create_coeffs2(double* input, Eigen::Matrix<double,1,10> M[6][4], int *Sub_index, int Sub_index_N,
                               int *Sub_index_extra, int Sub_index_extra_N, AC_TYPE actype, bool is_known_angle) {
    double C[1260];
    construct_order6_poly(M, C, Sub_index, Sub_index_N);

    double C_extra[168];
    if (actype == GENERIC_CONSTRAINT_FULL || actype == CASE5_CONSTRAINT_FULL 
        || actype == INTER_CAM_CONSTRAINT_FULL || actype == INTRA_CAM_CONSTRAINT_FULL) {
        construct_order6_extra_poly(M, C_extra, Sub_index_extra, Sub_index_extra_N);
    }

    // prepare data for Matlab interface
    // Matlab memory is column-major order
    int cnt = 0;
    if (actype == GENERIC_CONSTRAINT_PARTIAL || actype == CASE5_CONSTRAINT_PARTIAL 
        || actype == INTER_CAM_CONSTRAINT_PARTIAL || actype == INTRA_CAM_CONSTRAINT_PARTIAL) {
        for (int j = 0; j < 84; j++) {
            for (int i = 0; i < Sub_index_N; i++) {
                cnt++;
            }
        }
    }

    cnt = 0;
    if (actype == GENERIC_CONSTRAINT_FULL || actype == CASE5_CONSTRAINT_FULL 
        || actype == INTER_CAM_CONSTRAINT_FULL || actype == INTRA_CAM_CONSTRAINT_FULL) {
        for (int j = 0; j < 84; j++) {
            for (int i = 0; i < Sub_index_N; i++) {
                cnt++;
            }

            for (int i = 0; i < Sub_index_extra_N; i++) {
                cnt++;
            }
        }
    }

    // prepare data for the solver
    cnt = 0;
    if (actype == GENERIC_CONSTRAINT_PARTIAL || actype == CASE5_CONSTRAINT_PARTIAL
        ||actype == INTER_CAM_CONSTRAINT_PARTIAL) {
        for (int i = 0; i < Sub_index_N; i++) {
            for (int j = 0; j < 84; j++) {
                int idxxxx = Sub_index[i];
                input[cnt] = C[idxxxx*84+j];
                cnt++;
            }
        }
    }

    cnt = 0;
    if (actype == INTRA_CAM_CONSTRAINT_PARTIAL) {
        for (int i = 0; i < Sub_index_N; i++) {
            for (int j = 0; j < 83; j++) {
                int idxxxx = Sub_index[i];
                input[cnt] = C[idxxxx * 84 + j];
                cnt++;
            }
        }
    }

    cnt = 0;
    if (actype == GENERIC_CONSTRAINT_FULL || actype == CASE5_CONSTRAINT_FULL
        || actype == INTER_CAM_CONSTRAINT_FULL) {
        for (int i = 0; i < Sub_index_N; i++) {
            for (int j = 0; j < 84; j++) {
                int idxxxx = Sub_index[i];
                input[cnt] = C[idxxxx * 84 + j];
                cnt++;
            }
        }

        if (is_known_angle) {
            cnt+=4;
        }

        for (int i = 0; i < Sub_index_extra_N; i++) {
            for (int j = 49; j < 84; j++) {
                int idxxxx = Sub_index_extra[i];
                input[cnt] = C_extra[idxxxx*84+j];
                cnt++;
            }
        }
    }

    cnt = 0;
    if (actype == INTRA_CAM_CONSTRAINT_FULL) {
        for (int i = 0; i < Sub_index_N; i++) {
            for (int j = 0; j < 83; j++) {
                int idxxxx = Sub_index[i];
                input[cnt] = C[idxxxx * 84 + j];
                cnt++;
            }
        }

        if (is_known_angle) {
            cnt+=4;
        }

        for (int i = 0; i < Sub_index_extra_N; i++) {
            for (int j = 49; j < 84; j++) {
                int idxxxx = Sub_index_extra[i];
                input[cnt] = C_extra[idxxxx*84+j];
                cnt++;
            }
        }
    }
}

__device__ void create_coeffs(double* input, Eigen::Matrix<double,1,10> M[6][4],
    double* input_Image_1, double* input_Image_2, double* input_affine_tran,
    double* extrinsic_R_camera, double* extrinsic_T_camera, AC_TYPE actype, 
    bool is_known_angle) {

    if (actype == GENERIC_CONSTRAINT_FULL || actype == GENERIC_CONSTRAINT_PARTIAL) {
        Eigen::Matrix3d R_camera[4];
        Eigen::Vector3d T_camera[4];
        Eigen::Matrix3d Ac[2];
        Eigen::Vector3d Image1[2];
        Eigen::Vector3d Image2[2];
        format_convert_generic(input_Image_1, input_Image_2, input_affine_tran, extrinsic_R_camera, extrinsic_T_camera, Image1, Image2, Ac, R_camera, T_camera);
        construct_M(M, R_camera, T_camera, Ac, Image1, Image2, actype, is_known_angle);
    } else {
        Eigen::Matrix3d R_camera[2];
        Eigen::Vector3d T_camera[2];
        Eigen::Matrix3d Ac[2];
        Eigen::Vector3d Image1[2];
        Eigen::Vector3d Image2[2];
        format_convert(input_Image_1, input_Image_2, input_affine_tran, extrinsic_R_camera, extrinsic_T_camera, Image1, Image2, Ac, R_camera, T_camera);
        construct_M(M, R_camera, T_camera, Ac, Image1, Image2, actype, is_known_angle);
    }

    if (!is_known_angle) {
        int Sub_index_N = 15;
        int Sub_index_extra_N = 2;

        int Sub_index[15] = { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14 };
        int Sub_index_extra[2] = {0, 1};

        create_coeffs2(input, M, Sub_index, Sub_index_N, Sub_index_extra, Sub_index_extra_N, actype, is_known_angle);
    } else {
        int Sub_index_N = 5;
        int Sub_index_extra_N = 1;

        int Sub_index[5] = { 0, 1, 3, 6, 10 };
        int Sub_index_extra[1] = {0};

        create_coeffs2(input, M, Sub_index, Sub_index_N, Sub_index_extra, Sub_index_extra_N, actype, is_known_angle);
    }
}

__device__ void calculate_M_6_by_4(Eigen::Matrix<double, 6, 4>& M_double, Eigen::Matrix<double,1,10> M[6][4], double x, double y, double z) {    
    Eigen::Matrix<double, 10, 1> m;
    m(0) = x*x;
    m(1) = x*y;
    m(2) = x*z;
    m(3) = x;
    m(4) = y*y;
    m(5) = y*z;
    m(6) = y;
    m(7) = z*z;
    m(8) = z;
    m(9) = 1;

    for (int i = 0; i < 6; i++) {
        for (int j = 0; j < 4; j++) {
            Eigen::Matrix<double,1,1> rslt= M[i][j]*m;
            M_double(i,j) = rslt(0);
        }
    }

    return;
}

__device__ void calculate_translation(
    Eigen::MatrixXd sols, Eigen::Matrix<double,1,10> M[6][4], 
    Eigen::Matrix<double,3,1>* q_arr, Eigen::Matrix<double,3,1>* t_arr, bool is_known_angle) {

    for (int s = 0; s < sols.cols(); s++) {
        double x, y, z;
        x = sols(0, s);
        y = sols(1, s);
        z = sols(2, s);
        
        Eigen::Vector3d q;
        q << x, y, z;

        q_arr[s] = q;

        Eigen::Matrix<double, 6, 4> M_double;
        calculate_M_6_by_4(M_double, M, x, y, z);

        Eigen::Matrix<double, 3, 1> C12;
        if (is_known_angle) {
            Eigen::Matrix<double, 5, 3> C0 = M_double.block(0, 0, 5, 3);
            Eigen::Matrix<double, 5, 1> C1 = M_double.block(0, 3, 5, 1);
            // C12 = -C0.colPivHouseholderQr().solve(C1);
            double A[5*3];
            double B[5*1];
            double X[3*1];

            for (int i = 0; i < 5; i++) 
                for (int j = 0; j < 3; j++) 
                    A[i*3+j] = C0(i, j);

            for (int i = 0; i < 5; i++) 
                for (int j = 0; j < 1; j++) 
                    B[i*1+j] = C1(i, j);

            double QR[5*3];
            double RDiag[3];
            double X1[5*1];
            qr::solve2(5, 3, 1, 3, A, B, X, QR, RDiag, X1);
            for (int i = 0; i < 3; i++) 
                for (int j = 0; j < 1; j++) 
                    C12(i, j) = -X[i*1+j];
        } else {
            Eigen::Matrix<double, 6, 3> C0 = M_double.block(0, 0, 6, 3);
            Eigen::Matrix<double, 6, 1> C1 = M_double.block(0, 3, 6, 1);
            // C12 = -C0.colPivHouseholderQr().solve(C1);
            double A[6*3];
            double B[6*1];
            double X[3*1];

            for (int i = 0; i < 6; i++) 
                for (int j = 0; j < 3; j++) 
                    A[i*3+j] = C0(i, j);

            for (int i = 0; i < 6; i++) 
                for (int j = 0; j < 1; j++) 
                    B[i*1+j] = C1(i, j);

            double QR[6*3];
            double RDiag[3];
            double X1[6*1];
            qr::solve2(6, 3, 1, 3, A, B, X, QR, RDiag, X1);
            for (int i = 0; i < 3; i++) 
                for (int j = 0; j < 1; j++) 
                    C12(i, j) = -X[i*1+j];
        }
        
        t_arr[s] = C12;
    }

    return;
}