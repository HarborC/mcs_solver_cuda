#include "hip/hip_runtime.h"
#include "qr_cuda.h"
#include "utils_cuda.h"

// #include <iostream>
// #include <cmath>
// using namespace std;

namespace qr {
	
__device__ void dec(int m, int n, int p, double **A, double** QR, double* RDiag) {
	// main loop.
	for (int k = 0; k<p; ++k) {
		// Compute 2-norm of k-th column without under/overflow.
		double nrm = 0;
		for (int i = k; i<m; ++i)
			nrm = hypot(nrm, QR[i][k]);
		//        for( int i=k; i<m; ++i )
		//            nrm += QR[i][k]*QR[i][k];
		//        nrm = sqrt(nrm);

		if (nrm != 0) {
			// Form k-th Householder vector.
			if (QR[k][k] < 0)
				nrm = -nrm;

			for (int i = k; i<m; ++i)
				QR[i][k] /= nrm;

			QR[k][k] += 1;

			// Apply transformation to remaining columns.
			for (int j = k + 1; j<n; ++j) {
				double s = 0;
				for (int i = k; i<m; ++i)
					s += QR[i][k] * QR[i][j];

				s = -s / QR[k][k];
				for (int i = k; i<m; ++i)
					QR[i][j] += s*QR[i][k];
			}
		}

		RDiag[k] = -nrm;
	}
}

__device__ bool isFullRank(int p, double* RDiag) {
	for (int j = 0; j<p; ++j)
	if (RDiag[j] == 0)
		return false;

	return true;
}

__device__ bool solve(int m, int n, double** A, double *b, double* x_) {
	int p = min(m, n);

	double** QR;
	double* RDiag;
	malloc_matrix(m, n, &QR);
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			QR[i][j] = A[i][j];
		}
	}
	malloc_vector(p, &RDiag);

	dec(m, n, p, A, QR, RDiag);

	// matrix is rank deficient
	if (!isFullRank(p, RDiag)) {
		free_matrix(m, n, QR);
		free_vector(p, RDiag);
		return false;
	}

	double* x;
	malloc_vector(n, &x);
	for (int i = 0; i<n; ++i)
		x[i] = b[i];

	// compute y = transpose(Q)*b
	for (int k = 0; k<n; ++k) {
		double s = 0;
		for (int i = k; i<m; ++i)
			s += QR[i][k] * x[i];

		s = -s / QR[k][k];
		for (int i = k; i<m; ++i)
			x[i] += s*QR[i][k];
	}

	// solve R*x = y;
	for (int k = n - 1; k >= 0; --k) {
		x[k] /= RDiag[k];
		for (int i = 0; i<k; ++i)
			x[i] -= x[k] * QR[i][k];
	}

	// return n portion of x
	for (int i = 0; i<n; ++i)
		x_[i] = x[i];

	free_vector(n, x);
	free_matrix(m, n, QR);
	free_vector(p, RDiag);
	return true;
}

__device__ bool solve(int m, int n, int nx, double** A, double **B, double** X_) {
	int p = min(m, n);

	double** QR;
	double* RDiag;
	malloc_matrix(m, n, &QR);
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			QR[i][j] = A[i][j];
		}
	}
	malloc_vector(p, &RDiag);

	dec(m, n, p, A, QR, RDiag);

	// matrix is rank deficient
	if (!isFullRank(p, RDiag)) {
		free_matrix(m, n, QR);
		free_vector(p, RDiag);
		return false;
	}

	double** X;
	malloc_matrix(m, nx, &X);
	for (int i = 0; i<m; ++i) {
		for (int j = 0; j<nx; ++j) {
			X[i][j] = B[i][j];
		}
	}
	
	// compute Y = transpose(Q)*B
	for (int k = 0; k<n; ++k)
		for (int j = 0; j<nx; ++j) {
			double s = 0;
			for (int i = k; i<m; ++i)
				s += QR[i][k] * X[i][j];

			s = -s / QR[k][k];
			for (int i = k; i<m; ++i)
				X[i][j] += s*QR[i][k];
		}

	// solve R*X = Y;
	for (int k = n - 1; k >= 0; --k) {
		for (int j = 0; j<nx; ++j)
			X[k][j] /= RDiag[k];

		for (int i = 0; i<k; ++i)
			for (int j = 0; j<nx; ++j)
				X[i][j] -= X[k][j] * QR[i][k];
	}

	// return n x nx portion of X
	// Matrix<double> X_(n, nx);
	for (int i = 0; i<n; ++i)
		for (int j = 0; j<nx; ++j)
			X_[i][j] = X[i][j];

	free_matrix(m, nx, X);
	free_matrix(m, n, QR);
	free_vector(p, RDiag);

	return true;
}

__device__ void test() {
	int m = 4;
    int n = 3;
    int nx = 4;
    double** A;
    double** B;
    double** X;

    double A_vec[] = {1,0,0,
					  1,2,4,
					  1,3,9,
					  1,3,9};

    malloc_matrix(m, n, &A);
	memcopy_matrix(m, n, A, &(A_vec[0]));

    double B_vec[] = {1,0,0,0,
                      0,1,0,0,
					  0,0,1,0,
					  0,0,0,1};
    malloc_matrix(m, nx, &B);
	memcopy_matrix(m, nx, B, &B_vec[0]);

    malloc_matrix(n, nx, &X);
    solve(m, n, nx, A, B, X);

    print_matrix(n, nx, X);
}

}