#include "hip/hip_runtime.h"
#include "evd_cuda.h"
#include "utils_cuda.h"

// #include <iostream>
// #include <cmath>
// #include <vector>
// using namespace std;

namespace evd {

__device__ void tred2(int n, double** V, double* d, double* e) {
	for (int j = 0; j < n; ++j)
		d[j] = V[n - 1][j];

	// Householder reduction to tridiagonal form
	for (int i = n - 1; i>0; --i) {
		// scale to avoid under/overflow
		double scale = 0;
		double h = 0;

		for (int k = 0; k<i; ++k)
			scale += abs(d[k]);

		if (scale == 0) {
			e[i] = d[i - 1];
			for (int j = 0; j<i; ++j) {
				d[j] = V[i - 1][j];
				V[i][j] = 0;
				V[j][i] = 0;
			}
		} else {
			// generate Householder vector
			for (int k = 0; k<i; ++k) {
				d[k] /= scale;
				h += d[k] * d[k];
			}

			double f = d[i - 1];
			double g = sqrt(h);
			if (f > 0)
				g = -g;

			e[i] = scale * g;
			h = h - f * g;
			d[i - 1] = f - g;

			for (int j = 0; j<i; ++j)
				e[j] = 0;

			// Apply similarity transformation to remaining columns.
			for (int j = 0; j<i; ++j) {
				f = d[j];
				V[j][i] = f;
				g = e[j] + V[j][j] * f;

				for (int k = j + 1; k <= i - 1; ++k) {
					g += V[k][j] * d[k];
					e[k] += V[k][j] * f;
				}
				e[j] = g;
			}

			f = 0;
			for (int j = 0; j<i; ++j) {
				e[j] /= h;
				f += e[j] * d[j];
			}

			double hh = f / (h + h);
			for (int j = 0; j<i; ++j)
				e[j] -= hh * d[j];

			for (int j = 0; j<i; ++j) {
				f = d[j];
				g = e[j];
				for (int k = j; k <= i - 1; ++k)
					V[k][j] -= (f*e[k] + g*d[k]);

				d[j] = V[i - 1][j];
				V[i][j] = 0;
			}
		}
		d[i] = h;
	}

	// accumulate transformations
	for (int i = 0; i<n - 1; i++) {
		V[n - 1][i] = V[i][i];
		V[i][i] = 1;
		double h = d[i + 1];

		if (h != 0) {
			for (int k = 0; k <= i; ++k)
				d[k] = V[k][i + 1] / h;

			for (int j = 0; j <= i; ++j) {
				double g = 0;
				for (int k = 0; k <= i; ++k)
					g += V[k][i + 1] * V[k][j];

				for (int k = 0; k <= i; ++k)
					V[k][j] -= g * d[k];
			}
		}

		for (int k = 0; k <= i; ++k)
			V[k][i + 1] = 0;
	}

	for (int j = 0; j<n; ++j) {
		d[j] = V[n - 1][j];
		V[n - 1][j] = 0;
	}

	V[n - 1][n - 1] = 1;
	e[0] = 0;
}

__device__ void tql2(int n, double** V, double* d, double* e) {
	for (int i = 1; i<n; ++i)
		e[i - 1] = e[i];
	e[n - 1] = 0;

	double f = 0;
	double tst1 = 0;
	double eps = pow(2.0, -52.0);

	for (int l = 0; l<n; ++l) {
		// find small subdiagonal element
		tst1 = max(tst1, abs(d[l]) + abs(e[l]));
		int m = l;

		// original while-loop from Java code
		while (m < n) {
			if (abs(e[m]) <= eps*tst1)
				break;
			m++;
		}

		// if m == l, d[l] is an eigenvalue, otherwise, iterate
		if (m > l) {
			int iter = 0;
			do {
				iter = iter + 1;

				// compute implicit shift
				double g = d[l];
				double p = (d[l + 1] - g) / (2.0 * e[l]);
				double r = hypot(p, 1.0);
				if (p < 0)
					r = -r;

				d[l] = e[l] / (p + r);
				d[l + 1] = e[l] * (p + r);
				double dl1 = d[l + 1];
				double h = g - d[l];

				for (int i = l + 2; i<n; ++i)
					d[i] -= h;
				f += h;

				// implicit QL transformation.
				p = d[m];
				double c = 1;
				double c2 = c;
				double c3 = c;
				double el1 = e[l + 1];
				double s = 0;
				double s2 = 0;

				for (int i = m - 1; i >= l; --i) {
					c3 = c2;
					c2 = c;
					s2 = s;
					g = c * e[i];
					h = c * p;
					r = hypot(p, e[i]);
					e[i + 1] = s * r;
					s = e[i] / r;
					c = p / r;
					p = c * d[i] - s * g;
					d[i + 1] = h + s * (c * g + s * d[i]);

					// accumulate transformation.
					for (int k = 0; k<n; ++k) {
						h = V[k][i + 1];
						V[k][i + 1] = s * V[k][i] + c * h;
						V[k][i] = c * V[k][i] - s * h;
					}
				}

				p = -s * s2 * c3 * el1 * e[l] / dl1;
				e[l] = s * p;
				d[l] = c * p;

			} while (abs(e[l]) > eps*tst1);
		}

		d[l] += f;
		e[l] = 0;
	}

	// Sort eigenvalues and corresponding vectors.
	for (int i = 0; i<n - 1; ++i) {
		int k = i;
		double p = d[i];

		for (int j = i + 1; j<n; ++j)
		if (d[j] < p) {
			k = j;
			p = d[j];
		}

		if (k != i) {
			d[k] = d[i];
			d[i] = p;
			for (int j = 0; j<n; ++j) {
                double t = V[j][k];
                V[j][i] = V[j][k];
                V[j][k] = t;
            }
		}
	}
}

__device__ void others(int n, double** V, double **H, double *ort) {
	int low = 0;
	int high = n - 1;

	for (int m = low + 1; m <= high - 1; ++m) {
		// scale column.
		double scale = 0;
		for (int i = m; i <= high; ++i)
			scale += abs(H[i][m - 1]);

		if (scale != 0) {
			// compute Householder transformation.
			double h = 0;
			for (int i = high; i >= m; --i) {
				ort[i] = H[i][m - 1] / scale;
				h += ort[i] * ort[i];
			}

			double g = sqrt(h);
			if (ort[m] > 0)
				g = -g;

			h = h - ort[m] * g;
			ort[m] = ort[m] - g;

			// Apply Householder similarity transformation.
			for (int j = m; j<n; ++j) {
				double f = 0;
				for (int i = high; i >= m; --i)
					f += ort[i] * H[i][j];
				f = f / h;

				for (int i = m; i <= high; ++i)
					H[i][j] -= f*ort[i];
			}

			for (int i = 0; i <= high; ++i) {
				double f = 0;
				for (int j = high; j >= m; --j)
					f += ort[j] * H[i][j];
				f = f / h;

				for (int j = m; j <= high; ++j)
					H[i][j] -= f*ort[j];
			}

			ort[m] = scale * ort[m];
			H[m][m - 1] = scale * g;
		}
	}

	// accumulate transformations (Algol's ortran)
	for (int i = 0; i<n; ++i)
        for (int j = 0; j<n; ++j)
            V[i][j] = (i == j) ? 1 : 0;

	for (int m = high - 1; m >= low + 1; --m)
        if (H[m][m - 1] != 0) {
            for (int i = m + 1; i <= high; ++i)
                ort[i] = H[i][m - 1];

            for (int j = m; j <= high; ++j) {
                double g = 0;
                for (int i = m; i <= high; ++i)
                    g += ort[i] * V[i][j];

                // double division avoids possible underflow
                g = (g / ort[m]) / H[m][m - 1];
                for (int i = m; i <= high; ++i)
                    V[i][j] += g * ort[i];
            }
        }
}

__device__ void cdiv(double xr, double xi, double yr, double yi, double* cdivr, double* cdivi) {
	double r, d;
	if (abs(yr) > abs(yi)) {
		r = yi / yr;
		d = yr + r*yi;
		*cdivr = (xr + r*xi) / d;
		*cdivi = (xi - r*xr) / d;
	} else {
		r = yr / yi;
		d = yi + r*yr;
		*cdivr = (r*xr + xi) / d;
		*cdivi = (r*xi - xr) / d;
	}
}

__device__ void hqr2(int n1, double** V, double* d, double* e, double **H, double *ort) {
    double cdivr, cdivi;

	// initialize
	int nn = n1;
	int n = nn - 1;
	int low = 0;
	int high = nn - 1;
	double eps = pow(2.0, -52.0);
	double exshift = 0;
	double p = 0, q = 0, r = 0, s = 0, z = 0, t, w, x, y;

	// Store roots isolated by balanc and compute matrix norm.
	double norm = 0;
	for (int i = 0; i<nn; ++i) {
		if ((i < low) || (i > high)) {
			d[i] = H[i][i];
			e[i] = 0;
		}

		for (int j = max(i - 1, 0); j<nn; ++j)
			norm += abs(H[i][j]);
	}

	// outer loop over eigenvalue index
	int iter = 0;
	while (n >= low) {
		// Look for single small sub-diagonal element.
		int l = n;
		while (l > low) {
			s = abs(H[l - 1][l - 1]) + abs(H[l][l]);
			if (s == 0)
				s = norm;

			if (abs(H[l][l - 1]) < eps*s)
				break;

			l--;
		}

		// one root found
		if (l == n) {
			H[n][n] = H[n][n] + exshift;
			d[n] = H[n][n];
			e[n] = 0;
			n--;
			iter = 0;
		}
		// two roots found
		else if (l == n - 1) {
			w = H[n][n - 1] * H[n - 1][n];
			p = (H[n - 1][n - 1] - H[n][n]) / 2.0;
			q = p * p + w;
			z = sqrt(abs(q));
			H[n][n] = H[n][n] + exshift;
			H[n - 1][n - 1] = H[n - 1][n - 1] + exshift;
			x = H[n][n];

			// real pair
			if (q >= 0) {
				if (p >= 0)
					z = p + z;
				else
					z = p - z;

				d[n - 1] = x + z;
				d[n] = d[n - 1];
				if (z != 0)
					d[n] = x - w / z;

				e[n - 1] = 0;
				e[n] = 0;
				x = H[n][n - 1];
				s = abs(x) + abs(z);
				p = x / s;
				q = z / s;
				r = sqrt(p * p + q * q);
				p = p / r;
				q = q / r;

				// row modification
				for (int j = n - 1; j<nn; ++j) {
					z = H[n - 1][j];
					H[n - 1][j] = q * z + p * H[n][j];
					H[n][j] = q * H[n][j] - p * z;
				}

				// column modification
				for (int i = 0; i <= n; ++i) {
					z = H[i][n - 1];
					H[i][n - 1] = q * z + p * H[i][n];
					H[i][n] = q * H[i][n] - p * z;
				}

				// accumulate transformations
				for (int i = low; i <= high; ++i) {
					z = V[i][n - 1];
					V[i][n - 1] = q * z + p * V[i][n];
					V[i][n] = q * V[i][n] - p * z;
				}
			}
			// complex pair
			else {
				d[n - 1] = x + p;
				d[n] = x + p;
				e[n - 1] = z;
				e[n] = -z;
			}

			n = n - 2;
			iter = 0;
		} else {

			// form shift
			x = H[n][n];
			y = 0;
			w = 0;

			if (l < n) {
				y = H[n - 1][n - 1];
				w = H[n][n - 1] * H[n - 1][n];
			}

			// Wilkinson's original ad hoc shift
			if (iter == 10) {
				exshift += x;
				for (int i = low; i <= n; ++i)
					H[i][i] -= x;

				s = abs(H[n][n - 1]) + abs(H[n - 1][n - 2]);
				x = y = 0.75 * s;
				w = -0.4375 * s * s;
			}

			// MATLAB's new ad hoc shift
			if (iter == 30) {
				s = (y - x) / 2.0;
				s = s * s + w;
				if (s > 0) {
					s = sqrt(s);
					if (y < x)
						s = -s;

					s = x - w / ((y - x) / 2.0 + s);
					for (int i = low; i <= n; ++i)
						H[i][i] -= s;

					exshift += s;
					x = y = w = 0.964;
				}
			}

			iter = iter + 1;

			// Look for two consecutive small sub-diagonal elements.
			int m = n - 2;
			while (m >= l) {
				z = H[m][m];
				r = x - z;
				s = y - z;
				p = (r * s - w) / H[m + 1][m] + H[m][m + 1];
				q = H[m + 1][m + 1] - z - r - s;
				r = H[m + 2][m + 1];
				s = abs(p) + abs(q) + abs(r);
				p = p / s;
				q = q / s;
				r = r / s;

				if (m == l)
					break;

				if (abs(H[m][m - 1]) * (abs(q) + abs(r)) <
					eps * (abs(p) * (abs(H[m - 1][m - 1]) + abs(z) +
					abs(H[m + 1][m + 1]))))
					break;

				m--;
			}

			for (int i = m + 2; i <= n; ++i) {
				H[i][i - 2] = 0;
				if (i > m + 2)
					H[i][i - 3] = 0;
			}

			// double QR step involving rows l:n and columns m:n
			for (int k = m; k <= n - 1; ++k) {
				int notlast = (k != n - 1);
				if (k != m) {
					p = H[k][k - 1];
					q = H[k + 1][k - 1];
					r = (notlast ? H[k + 2][k - 1] : 0);
					x = abs(p) + abs(q) + abs(r);

					if (x != 0) {
						p = p / x;
						q = q / x;
						r = r / x;
					}
				}

				if (x == 0)
					break;

				s = sqrt(p * p + q * q + r * r);
				if (p < 0)
					s = -s;

				if (s != 0) {
					if (k != m)
						H[k][k - 1] = -s * x;
					else if (l != m)
						H[k][k - 1] = -H[k][k - 1];

					p = p + s;
					x = p / s;
					y = q / s;
					z = r / s;
					q = q / p;
					r = r / p;

					// row modification
					for (int j = k; j<nn; ++j) {
						p = H[k][j] + q * H[k + 1][j];
						if (notlast) {
							p = p + r * H[k + 2][j];
							H[k + 2][j] = H[k + 2][j] - p * z;
						}

						H[k][j] = H[k][j] - p * x;
						H[k + 1][j] = H[k + 1][j] - p * y;
					}

					// column modification
					for (int i = 0; i <= min(n, k + 3); ++i) {
						p = x * H[i][k] + y * H[i][k + 1];
						if (notlast) {
							p = p + z * H[i][k + 2];
							H[i][k + 2] = H[i][k + 2] - p * r;
						}
						H[i][k] = H[i][k] - p;
						H[i][k + 1] = H[i][k + 1] - p * q;
					}

					// accumulate transformations
					for (int i = low; i <= high; ++i) {
						p = x * V[i][k] + y * V[i][k + 1];
						if (notlast)
						{
							p = p + z * V[i][k + 2];
							V[i][k + 2] = V[i][k + 2] - p * r;
						}
						V[i][k] = V[i][k] - p;
						V[i][k + 1] = V[i][k + 1] - p * q;
					}
				}  // (s != 0 )
			}  // k loop
		}  // check convergence
	}  // while ( n >= low )

	// Backsubstitute to find vectors of upper triangular form.
	if (norm == 0)
		return;

	for (n = nn - 1; n >= 0; --n) {
		p = d[n];
		q = e[n];

		// real vector
		if (q == 0) {
			int l = n;
			H[n][n] = 1;
			for (int i = n - 1; i >= 0; --i) {
				w = H[i][i] - p;
				r = 0;
				for (int j = l; j <= n; ++j)
					r = r + H[i][j] * H[j][n];

				if (e[i] < 0) {
					z = w;
					s = r;
				} else {
					l = i;
					if (e[i] == 0) {
						if (w != 0)
							H[i][n] = -r / w;
						else
							H[i][n] = -r / (eps * norm);
					}
					// solve real equations
					else {
						x = H[i][i + 1];
						y = H[i + 1][i];
						q = (d[i] - p) * (d[i] - p) + e[i] * e[i];
						t = (x * s - z * r) / q;
						H[i][n] = t;

						if (abs(x) > abs(z))
							H[i + 1][n] = (-r - w * t) / x;
						else
							H[i + 1][n] = (-s - y * t) / z;
					}

					// overflow control
					t = abs(H[i][n]);
					if ((eps*t)*t > 1)
						for (int j = i; j <= n; ++j)
							H[j][n] = H[j][n] / t;
				}
			}
		}
		// complex vector
		else if (q < 0) {
			int l = n - 1;

			// last vector component imaginary so matrix is triangular
			if (abs(H[n][n - 1]) > abs(H[n - 1][n])) {
				H[n - 1][n - 1] = q / H[n][n - 1];
				H[n - 1][n] = -(H[n][n] - p) / H[n][n - 1];
			} else {
				cdiv(0, -H[n - 1][n], H[n - 1][n - 1] - p, q, &cdivr, &cdivi);
				H[n - 1][n - 1] = cdivr;
				H[n - 1][n] = cdivi;
			}

			H[n][n - 1] = 0;
			H[n][n] = 1;
			for (int i = n - 2; i >= 0; --i) {
				double ra, sa, vr, vi;
				ra = 0;
				sa = 0;
				for (int j = l; j <= n; ++j) {
					ra = ra + H[i][j] * H[j][n - 1];
					sa = sa + H[i][j] * H[j][n];
				}
				w = H[i][i] - p;

				if (e[i] < 0) {
					z = w;
					r = ra;
					s = sa;
				} else {
					l = i;
					if (e[i] == 0) {
						cdiv(-ra, -sa, w, q, &cdivr, &cdivi);
						H[i][n - 1] = cdivr;
						H[i][n] = cdivi;
					} else {
						// solve complex equations
						x = H[i][i + 1];
						y = H[i + 1][i];
						vr = (d[i] - p) * (d[i] - p) + e[i] * e[i] - q*q;
						vi = (d[i] - p) * 2.0 * q;
						if ((vr == 0) && (vi == 0))
							vr = eps * norm * (abs(w) + abs(q) +
							abs(x) + abs(y) + abs(z));

						cdiv(x*r - z*ra + q*sa, x*s - z*sa - q*ra, vr, vi, &cdivr, &cdivi);
						H[i][n - 1] = cdivr;
						H[i][n] = cdivi;

						if (abs(x) > (abs(z) + abs(q))) {
							H[i + 1][n - 1] = (-ra - w*H[i][n - 1] + q*H[i][n]) / x;
							H[i + 1][n] = (-sa - w*H[i][n] - q*H[i][n - 1]) / x;
						} else {
							cdiv(-r - y*H[i][n - 1], -s - y*H[i][n], z, q, &cdivr, &cdivi);
							H[i + 1][n - 1] = cdivr;
							H[i + 1][n] = cdivi;
						}
					}

					// overflow control
					t = max(abs(H[i][n - 1]), abs(H[i][n]));
					if ((eps*t)*t > 1)
						for (int j = i; j <= n; ++j) {
							H[j][n - 1] = H[j][n - 1] / t;
							H[j][n] = H[j][n] / t;
						}
				}
			}
		}
	}

	// vectors of isolated roots
	for (int i = 0; i<nn; ++i)
		if ((i < low) || (i > high))
			for (int j = i; j<nn; ++j)
				V[i][j] = H[i][j];

	// Back transformation to get eigenvectors of original matrix.
	for (int j = nn - 1; j >= low; --j)
		for (int i = low; i <= high; ++i) {
			z = 0;
			for (int k = low; k <= min(j, high); ++k)
				z += V[i][k] * H[k][j];

			V[i][j] = z;
		}
}

__device__ void getRealV(int n, double** V, double* d) {
	double** cV;
	malloc_matrix(n, n, &cV);
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			cV[i][j] = V[i][j];
		}
	}

	int col = 0;
	while (col < n - 1) {
		// eigenvalues d[col] and d[col+1] are complex
		if (d[col] == d[col + 1]) {
			for (int i = 0; i<n; ++i) {
				cV[i][col] = V[i][col];
				cV[i][col + 1] = cV[i][col];
			}
			col += 2;
		}
		// eigenvalue d[col] is real
		else {
			for (int i = 0; i<n; ++i)
				cV[i][col] = V[i][col];
			col += 1;
		}
	}

	// eigenvalue d[n-1] is real
	if (col == n - 1) {
		for (int i = 0; i<n; ++i)
			cV[i][col] = V[i][col];
		col += 1;
	}

	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			V[i][j] = cV[i][j];
		}
	}

	free_matrix(n, n, cV);
}

__device__ void dec(int n, double** A, double** V, double* d) {

	// V = Matrix<double>(n, n);
	// d = Vector<double>(n);
	// e = Vector<double>(n);
    double* e;
	malloc_vector(n, &e);

	bool symmetric = true;
	for (int j = 0; (j<n) && symmetric; ++j)
        for (int i = 0; (i<n) && symmetric; ++i)
            symmetric = (A[i][j] == A[j][i]);

	if (symmetric) {
		for (int i = 0; i<n; ++i)
            for (int j = 0; j<n; ++j)
                V[i][j] = A[i][j];

		// tridiagonalize.
		tred2(n, V, d, e);

		// diagonalize.
		tql2(n, V, d, e);
	} else {
		double **H; // = Matrix<double>(n, n);
		double* ort; // = Vector<double>(n);
		malloc_matrix(n, n, &H);
		malloc_vector(n, &ort);

		for (int j = 0; j<n; ++j)
            for (int i = 0; i<n; ++i)
                H[i][j] = A[i][j];

		// reduce to Hessenberg form
		others(n, V, H, ort);

		// print_matrix(n, n, H);
		
		// reduce Hessenberg to real Schur form
		hqr2(n, V, d, e, H, ort);

		// print_matrix(n, n, H);

		free_matrix(n, n, H);
		free_vector(n, ort);
	}

	getRealV(n, V, d);

	free_vector(n, e);
}

// __device__ void test() {
//     int n = 7;
//     double** A;
//     double** V;
//     double* d;

// 	double A_vec[] = {3,4,5,10,3,7,1,
// 					  7,9,2,6,2,4,8,
// 					  5,2,4,7,1,9,4,
// 					  12,78,32,65,31,23,16,
// 					  95,74,36,21,28,49,58,
// 					  12,18,16,17,34,52,29,
// 					  2,7,99,31,57,90,83};
// 	malloc_matrix(n, n, &A);
// 	memcopy_matrix(n, n, A, &(A_vec[0]));
// 	malloc_matrix(n, n, &V);
// 	malloc_vector(n, &d);

// 	evd::dec(n, A, V, d);

// 	print_matrix(n, n, A);
// 	print_matrix(n, n, V);
// 	print_vector(n, d);

// 	free_matrix(n, n, A);
// 	free_matrix(n, n, V);
// 	free_vector(n, d);
// }

}